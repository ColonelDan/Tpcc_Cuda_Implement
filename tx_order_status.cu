#include <stdio.h>

#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>

#include "tpcc_table.h"
#include "tx.h"
#include "utility.h"
#include "table_operator.h"

#define MAX_PRINT_ORDER_LINE 1000

//test for order_status transaction!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!

__device__
void tx_order_status(){
	//输入的数据
	long os_C_W_ID=1;	long os_C_D_ID=3;	long os_C_ID=3;	//customer ID


	//事务启动
	printf("Order Status Transaction is start!\n");


	//在客户表（customers）中查询客户代码（C_W_ID,C_D_ID,C_ID）。
	struct customer customer_tmp;
	struct customer *p_customer_tmp;
	int rid_in_customers=0;
	while(1){
		rid_in_customers = table_scan(CUSTOMER, LONG, sizeof(long int), ((long)&(customer_tmp.C_W_ID)-(long)&(customer_tmp.C_ID)), EQ, &os_C_W_ID, rid_in_customers);	//扫描整张customer表找出C_W_ID为os_C_W_ID的记录号
		if(rid_in_customers == -1){	//customers中没有os_C_W_ID对应的记录 
			printf("Transaction Exception:\tC_W_ID is not in customer table!\n");
			return;
		}
		p_customer_tmp=(struct customer *)get(CUSTOMER, rid_in_customers);	//获取os_C_W_ID对应的记录
		if(p_customer_tmp->C_D_ID == os_C_D_ID && p_customer_tmp->C_ID == os_C_ID){	//customers中有os_C_W_ID,os_C_D_ID,os_C_ID对应的记录
			break;
		}else{
			rid_in_customers++;
		}
	}
	printf("rid_in_customers:%d\tC_W_ID:%ld\tC_D_ID:%ld\tC_ID:%ld\tC_FIRST:%s\tC_MIDDLE:%s\tC_LAST:%s\tC_BALANCE:%lf\n",
		rid_in_customers,
		p_customer_tmp->C_W_ID,
		p_customer_tmp->C_D_ID,
		p_customer_tmp->C_ID,
		p_customer_tmp->C_FIRST,	//取姓名
		p_customer_tmp->C_MIDDLE,
		p_customer_tmp->C_LAST,
		p_customer_tmp->C_BALANCE);	//欠款余额


	//在定单表（orders）中查询仓库代码（O_W_ID）、地区代码（O_D_ID）、客户代码（O_C_ID），并且定单代码（O_ID）最大的记录。
	struct order order_tmp;
	struct order *p_order_tmp;
	int rid_in_orders=0;
	int os_rid_in_orders_max=-1;
	long os_O_ID_max=-1;
	while(1){
		rid_in_orders = table_scan(ORDER, LONG, sizeof(long int), ((long)&(order_tmp.O_W_ID)-(long)&(order_tmp.O_ID)), EQ, &os_C_W_ID, rid_in_orders);	//扫描整张order表找出O_W_ID为os_C_W_ID的记录号
		if(rid_in_orders == -1){
			break;
		}
		p_order_tmp=(struct order *)get(ORDER, rid_in_orders);	//获取os_C_W_ID对应的记录
		if(p_order_tmp->O_D_ID == os_C_D_ID && p_order_tmp->O_C_ID == os_C_ID && p_order_tmp->O_ID > os_O_ID_max){
			os_rid_in_orders_max=rid_in_orders;
			os_O_ID_max=p_order_tmp->O_ID;
			rid_in_orders++;
		}else{
			rid_in_orders++;
		}
	}
	if(os_rid_in_orders_max == -1){	//orders中没有(os_C_W_ID、os_C_D_ID、os_C_ID)对应的记录 
		printf("Transaction Exception:\thave not found right record in order table!\n");
		return;
	}else{
		p_order_tmp=(struct order *)get(ORDER, os_rid_in_orders_max);
		printf("rid_in_orders:%d\tO_ID:%ld\tO_ENTRY_DATE:%ld\tO_CARRIER_ID:%ld\n", 
			os_rid_in_orders_max,
			p_order_tmp->O_ID,	//取定单代码
			p_order_tmp->O_ENTRY_DATE,	//制单日期
			p_order_tmp->O_CARRIER_ID);	//货运代码
	}


	//在定单分录表（orderlines）中查询满足仓库代码（OL_W_ID）、地区代码（OL_D_ID）、定单代码（OL_O_ID）条件的所有记录。
	struct order_line orderline_tmp;
	struct order_line *p_orderline_tmp;
	int rid_in_orderlines=0;
	struct order_line orderline_arr[MAX_PRINT_ORDER_LINE];
	int count=0;
	while(1){
		rid_in_orderlines = table_scan(ORDER_LINE, LONG, sizeof(long int), ((long)&(orderline_tmp.OL_W_ID)-(long)&(orderline_tmp.OL_O_ID)), EQ, &os_C_W_ID, rid_in_orderlines);	//扫描整张orderlines表找出C_W_ID为os_C_W_ID的记录号
		if(rid_in_orderlines == -1){
			break;
		}
		p_orderline_tmp=(struct order_line *)get(ORDER_LINE, rid_in_orderlines);	//获取os_C_W_ID对应的记录
		if(p_orderline_tmp->OL_D_ID == os_C_D_ID && p_orderline_tmp->OL_O_ID == p_order_tmp->O_ID){
			d_memcpy(&(orderline_arr[count]),p_orderline_tmp,sizeof(struct order_line));
			count++;
			if(count==MAX_PRINT_ORDER_LINE){
				break;
			}
			rid_in_orderlines++;
		}else{
			rid_in_orderlines++;
		}
	}
	if(count==0){
		printf("Transaction Exception:\thave not found right record in orderline table!\n");
		return;
	}else{
		int i,j,k;
		for(i=0;i<count;i++){	//根据商品代码排序并输出
			k=i;
			for(j=i+1;j<count;j++){
				if(orderline_arr[j].OL_I_ID>orderline_arr[k].OL_I_ID){
					k=j;
				}
			}
			if(i!=k){
				d_memcpy(&orderline_tmp,&orderline_arr[i],sizeof(struct order_line));
				d_memcpy(&orderline_arr[i],&orderline_arr[k],sizeof(struct order_line));
				d_memcpy(&orderline_arr[k],&orderline_tmp,sizeof(struct order_line));
			}
			printf("OL_I_ID:%ld\tOL_SUPPLY_W_ID:%ld\tOL_QUANTITY:%lf\tOL_AMOUNT:%lf\tOL_DELIVERY_D:%ld\n",
				orderline_arr[i].OL_I_ID,
				orderline_arr[i].OL_SUPPLY_W_ID,	//供应仓库代码
				orderline_arr[i].OL_QUANTITY,	//数量
				orderline_arr[i].OL_AMOUNT,	//金额
				orderline_arr[i].OL_DELIVERY_D);	//发货时间
		}
	}


	//事务提交 
	printf("Order Status Transaction is finished!\n");
}