#include <stdio.h>  
#include <string.h>  
#include <stdlib.h>

#define STOCK_RECORD_MAX_SIZE	300010 
#define STOCK_RECORD_MAX_LENGTH 1000
#include "tpcc_table.h"
int get_stock(struct stock *p_stock_arr){ //
	char file_name[] = "./csv/c_stock.csv";  //
    FILE *fp;  
    fp = fopen(file_name, "r");  
      
    if (!fp) {  
        fprintf(stderr, "c stock failed to open file for reading\n");  
        return -1;  
    }
    
    char c;
    int i=0;
    int index=0;
    
    char **record_arr;
    record_arr=(char **)malloc(STOCK_RECORD_MAX_SIZE*sizeof(char *));
    record_arr[index]=(char *)malloc(STOCK_RECORD_MAX_LENGTH*sizeof(char));
    
    while(1){
      c=fgetc(fp);
      if(c==EOF){
        break;
      }
      if(c=='\n'){
        record_arr[index][i++]='\0';
        index++;
        record_arr[index]=(char *)malloc(STOCK_RECORD_MAX_LENGTH*sizeof(char));
        i=0;
      }else{
        if(c!='"'){
          record_arr[index][i++]=c;
        }
      }
    }
    
    for(i=1;i<index;i++){
      char *token;
      token=strtok(record_arr[i],",");
      (p_stock_arr)[i].S_I_ID=atol(token);
     
      token=strtok(NULL, ",");
      (p_stock_arr)[i].S_W_ID=atol(token);//
	    token=strtok(NULL, ",");
      (p_stock_arr)[i].S_QUANTITY=atof(token);//
      
      token=strtok(NULL, ",");
      strcpy((p_stock_arr)[i].S_DIST_01,token);
      
      token=strtok(NULL, ",");
      strcpy((p_stock_arr)[i].S_DIST_02,token);

      token=strtok(NULL, ",");
      strcpy((p_stock_arr)[i].S_DIST_03,token);

      token=strtok(NULL, ",");
      strcpy((p_stock_arr)[i].S_DIST_04,token);

      token=strtok(NULL, ",");
      strcpy((p_stock_arr)[i].S_DIST_05,token);

      token=strtok(NULL, ",");
      strcpy((p_stock_arr)[i].S_DIST_06,token);

      token=strtok(NULL, ",");
      strcpy((p_stock_arr)[i].S_DIST_07,token);

      token=strtok(NULL, ",");
      strcpy((p_stock_arr)[i].S_DIST_08,token);

      token=strtok(NULL, ",");
      strcpy((p_stock_arr)[i].S_DIST_09,token);

      token=strtok(NULL, ",");
      strcpy((p_stock_arr)[i].S_DIST_10,token);

      token=strtok(NULL, ",");
      (p_stock_arr)[i].S_YTD=atof(token);//?????????????????????????????????
      
      token=strtok(NULL, ",");
      (p_stock_arr)[i].S_ORDER_CNT=atof(token);//?????????????????????????????????
      

      token=strtok(NULL, ",");
      (p_stock_arr)[i].S_REMOTE_CNT=atof(token);//?????????????????????????????????
      
      
	free(record_arr[i]);
    }
    free(record_arr);
    fclose (fp);  
    return index;
}
