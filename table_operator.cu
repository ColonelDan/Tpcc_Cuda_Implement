//
//  	desc: tpcc benchmark implementation in GPU
//	date: 2018-4-3
//	author: Xie Shangwei
//

#include <stdio.h>

#include <hip/hip_runtime.h>

#include "tpcc_table.h"
#include "utility.h"
#include "table_operator.h"

//get 9 tables.
extern struct warehouse h_warehouses[MAX_WAREHOUSE_NUM];
extern struct district h_districts[MAX_DISTRICT_NUM];
extern struct customer h_customers[MAX_CUSTOMER_NUM];
extern struct history h_historys[MAX_HISTORY_NUM];
extern struct new_order h_neworders[MAX_NEWORDER_NUM];
extern struct order h_orders[MAX_ORDER_NUM];
extern struct order_line h_orderlines[MAX_ORDER_LINE_NUM];
extern struct stock h_stocks[MAX_STOCK_NUM];
extern struct item h_items[MAX_ITEM_NUM];
 
//tables's slot flag, mark the slot is used or not.
extern char  h_warehouses_flag[MAX_WAREHOUSE_NUM];
extern char  h_districts_flag[MAX_DISTRICT_NUM];   
extern char  h_customers_flag[MAX_CUSTOMER_NUM];   
extern char  h_historys_flag[MAX_HISTORY_NUM];      
extern char  h_neworders_flag[MAX_NEWORDER_NUM];  
extern char  h_orders_flag[MAX_ORDER_NUM];            
extern char  h_orderlines_flag[MAX_ORDER_LINE_NUM];  
extern char  h_items_flag[MAX_ITEM_NUM];               
extern char  h_stocks_flag[MAX_STOCK_NUM];

extern struct warehouse *h_d_warehouses;
extern struct district *h_d_districts;
extern struct customer *h_d_customers;
extern struct history *h_d_historys;
extern struct new_order *h_d_new_orders;
extern struct order *h_d_orders;
extern struct order_line *h_d_orderlines;
extern struct item *h_d_items;
extern struct stock *h_d_stocks;

extern char *h_d_warehouses_flag;
extern char *h_d_districts_flag;
extern char *h_d_customers_flag;
extern char *h_d_historys_flag;
extern char *h_d_new_orders_flag;
extern char *h_d_orders_flag;
extern char *h_d_orderlines_flag;
extern char *h_d_items_flag;
extern char *h_d_stocks_flag;

extern __device__ char *d_warehouses_flag;
extern __device__ char *d_districts_flag;
extern __device__ char *d_customers_flag;
extern __device__ char *d_historys_flag;
extern __device__ char *d_new_orders_flag;
extern __device__ char *d_orders_flag;
extern __device__ char *d_orderlines_flag;
extern __device__ char *d_items_flag;
extern __device__ char *d_stocks_flag;

extern __device__ struct warehouse *d_warehouses;
extern __device__ struct district *d_districts;
extern __device__ struct customer *d_customers;
extern __device__ struct history *d_historys;
extern __device__ struct new_order *d_new_orders;
extern __device__ struct order *d_orders;
extern __device__ struct order_line *d_orderlines;
extern __device__ struct item *d_items;
extern __device__ struct stock *d_stocks;


__device__
void delete_rec(int table_type, int record_id){
	const int bid = blockIdx.x;
	const int tid = blockIdx.x;
	char *flag_head = NULL;
	if(bid == 0 && tid == 0){
		get_flag_head(table_type, &flag_head);
		mark_slot_free(flag_head, record_id);	
	}
}

__device__
int insert_rec(int table_type, void *record){
	int record_size = 0;
	int slot_id = -1;
	const int bid = blockIdx.x;
	const int tid = threadIdx.x;
	void *table_head= NULL;
	char *flag_head = NULL;
	if(bid == 0 && tid ==0){
		get_table_head(table_type, &table_head);
		get_record_size(table_type, &record_size);
		get_flag_head(table_type, &flag_head);
		slot_id = get_free_slot(flag_head, table_type);
		d_memcpy( table_head+slot_id*record_size, record, record_size);
		mark_slot_used(flag_head, slot_id);
			return slot_id;
	}
}

__device__
void update(int table_type, int record_id, void *record){
	void *table_head = NULL;
	int record_size;
	const int bid = blockIdx.x;
	const int tid = threadIdx.x;
	if( bid==0 && tid ==0){
		get_table_head(table_type, &table_head);
		get_record_size(table_type, &record_size);
		d_memcpy( table_head+record_id*record_size, record, record_size);
	}
}

__device__
void *get(int table_type, int record_id){
	void *table_head = NULL;
	int record_size;
	const int bid = blockIdx.x;
	const int tid = threadIdx.x;
	if(bid==0 && tid ==0){
		get_table_head(table_type, &table_head);
		get_record_size(table_type, &record_size);
		return (void *)(table_head+record_id*record_size);
	}
	return NULL;
}

// desc: scan table by compare some attribute in the table from the r_id.
// output: the record_id or -1 if there is no record match condition.
__device__
int table_scan(int table_type, int attr_type, int attr_size, int attr_offset, int op, void *value, int r_id){
	const int bid = blockIdx.x;
	const int tid = threadIdx.x;
	void *table_head = NULL;
	char *flag_head = NULL;
	int table_size;
	int record_size;
	if(bid == 0 && tid == 0){
		get_table_head(table_type, &table_head);
		get_table_size(table_type, &table_size);
		get_record_size(table_type, &record_size);
		get_flag_head(table_type, &flag_head);
		int i;
		for(i = r_id; i<table_size; i++){
			if(!(int)flag_head[i]){
				continue;
			}
			if(op == NO)
				return i;
			void *record_addr = (void *)(table_head + record_size*i);
			void *attr_addr = (void *)(record_addr + attr_offset);
			switch(attr_type){
				case(INT):
					{
					int des = *((int *)attr_addr);
					int src = *((int *)value);
					switch(op){
						case(EQ):
							if(des == src)
								return i;
							break;
						case(LT):
							if(des < src)
								return i;
							break;
						case(GT):
							if(des > src)
								return i;
							break;
						case(LE):
							if(des <= src)
								return i;
							break;
						case(GE):
							if(des >= src)
								return i;
							break;
						case(NE):
							if(des != src)
								return i;
							break;
					}
					}
					break;
				case(LONG):
					{
					long des = *((long *)attr_addr);
					long src = *((long *)value);
					switch(op){
						case(EQ):
							if(des == src)
								return i;
							break;
						case(LT):
							if(des < src)
								return i;
							break;
						case(GT):
							if(des > src)
								return i;
							break;
						case(LE):
							if(des <= src)
								return i;
							break;
						case(GE):
							if(des >= src)
								return i;
							break;
						case(NE):
							if(des != src)
								return i;
							break;
					}
					}
					break;
				case(DOUBLE):
					{
					double des = *((double *)attr_addr);
					double src = *((double *)value);
					switch(op){
						case(EQ):
							if(des == src)
								return i;
							break;
						case(LT):

							if(des < src)
								return i;
							break;
						case(GT):

							if(des > src)
								return i;
							break;
						case(LE):
							if(des <= src)
								return i;

							break;
						case(GE):
							if(des >= src)
								return i;
							break;
						case(NE):
							if(des != src)
								return i;
							break;
					}
					}
					break;
				case(STR):
					{
					char des[200];
					attr_size = d_strlen((char *)attr_addr);
					d_memcpy(des, (char *)attr_addr, attr_size);
					des[attr_size+1] = '\0';
					char src[200];
					attr_size = d_strlen((char *)value);
					d_memcpy(src, (char *)value, attr_size);
					src[attr_size+1] = '\0';
					//printf("des : %s src : %s\n", des, src);
					switch(op){
						case(EQ):
							if( !d_strcmp(des, src) ){
								//printf("equal\n");
								return i;
							}
							break;
						case(NE):
							if(d_strcmp(des, src)){
								//printf("not equal\n");
								return i;
							}
							break;
					}
					}
					break;
			}	
		}
	}	
	return -1;
}

/*
__device__
void get_next_record(){

}

__device__
void clsoe_scan(){

}
*/
__device__
void get_table_head(int table_type, void **table_head){

	switch(table_type){
		case(WAREHOUSE):
			*table_head = (void *)d_warehouses;
			break;
		case(STOCK):
			*table_head = (void *)d_stocks;
			break;
		case(DISTRICT):
			*table_head = (void *)d_districts;
			break;
		case(ITEM):
			*table_head = (void *)d_items;
			break;
		case(NEW_ORDER):
			*table_head = (void *)d_new_orders;
			break;
		case(ORDER):
			*table_head = (void *)d_orders;
			break;
		case(ORDER_LINE):
			*table_head = (void *)d_orderlines;
			break;
		case(CUSTOMER):
			*table_head = (void *)d_customers;
			break;
	}

}

__device__
void get_flag_head(int table_type, char **flag_head){

	switch(table_type){
		case(WAREHOUSE):
			*flag_head = d_warehouses_flag;
			break;
		case(STOCK):
			*flag_head = d_stocks_flag;
			break;
		case(DISTRICT):
			*flag_head = d_districts_flag;
			break;
		case(ITEM):
			*flag_head = d_items_flag;
			break;
		case(NEW_ORDER):
			*flag_head = d_new_orders_flag;
			break;
		case(ORDER):
			*flag_head = d_orders_flag;
			break;
		case(ORDER_LINE):
			*flag_head = d_orderlines_flag;
			break;
		case(CUSTOMER):
			*flag_head = d_customers_flag;
			break;
	}
}

__device__
void get_record_size(int table_type, int *record_size){
	
	switch(table_type){
		case(WAREHOUSE):
			*record_size = sizeof(struct warehouse);
			break;
		case(STOCK):
			*record_size = sizeof(struct stock);
			break;
		case(DISTRICT):
			*record_size = sizeof(struct district);
			break;
		case(ITEM):
			*record_size = sizeof(struct item);
			break;
		case(NEW_ORDER):
			*record_size = sizeof(struct new_order);
			break;
		case(ORDER):
			*record_size = sizeof(struct order);
			break;
		case(ORDER_LINE):
			*record_size = sizeof(struct order_line);
			break;
		case(CUSTOMER):
			*record_size = sizeof(struct customer);
			break;
	}
}

__device__
void get_table_size(int table_type, int *table_size){

	switch(table_type){
		case(WAREHOUSE):
			*table_size = MAX_WAREHOUSE_NUM;
			break;
		case(STOCK):
			*table_size = MAX_STOCK_NUM;
			break;
		case(DISTRICT):
			*table_size = MAX_DISTRICT_NUM;
			break;
		case(ITEM):
			*table_size = MAX_ITEM_NUM;
			break;
		case(NEW_ORDER):
			*table_size = MAX_NEWORDER_NUM;
			break;
		case(ORDER):
			*table_size = MAX_ORDER_NUM;
			break;
		case(ORDER_LINE):
			*table_size = MAX_ORDER_LINE_NUM;
			break;
		case(CUSTOMER):
			*table_size = MAX_CUSTOMER_NUM;
			break;
	}
}
