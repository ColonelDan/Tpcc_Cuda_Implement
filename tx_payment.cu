#include <stdio.h>

#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>

#include "tpcc_table.h"
#include "tx.h"
#include "utility.h"
#include "table_operator.h"

//test for payment transaction!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!

__device__
void tx_payment(){
	//输入的数据
	long pay_W_ID=1;	//warehouse ID
	long pay_D_W_ID=1;	long pay_D_ID=3;	//district ID
	long pay_C_W_ID=1;	long pay_C_D_ID=3;	long pay_C_ID=3;	//customer ID
	double pay_H_AMOUNT=52.0;	//支付金额


	//事务启动
	printf("Payment Transaction is start!\n");


	//在仓库表（warehouses）中查询仓库代码（W_ID）。取名称（W_NAME）、地址1（W_STREET_1）、地址2（W_STREET_2）、城市（W_CITY）、州（W_STATE）、邮政编码（W_ZIP）、累计发生额（W_YTD）。
	struct warehouse warehouse_tmp;	//为了求attr_offset 
	int rid_in_warehouses=0;
	//int table_scan(int table_type, int attr_type, int attr_size, int attr_offset, int op, void *value, int r_id);
	rid_in_warehouses= table_scan(WAREHOUSE, LONG, sizeof(long int), ((long)&(warehouse_tmp.W_ID)-(long)&(warehouse_tmp)), EQ, &pay_W_ID, rid_in_warehouses);	//扫描整张warehouses表找出W_ID为pay_W_ID的记录号 
	if(rid_in_warehouses==-1){	//warehouses中没有pay_W_ID对应的记录 
		printf("Transaction Exception:\tW_ID is not in warehouse table!");
		return;
	}
	struct warehouse *p_warehouse_tmp =(struct warehouse *)get(WAREHOUSE, rid_in_warehouses);	//获取no_W_ID对应的记录
	printf("rid_in_warehouses:%d\tW_ID:%ld\tW_NAME:%s\tW_STREET_1:%s\tW_STREET_2:%s\tW_CITY:%s\tW_STATE:%s\tW_ZIP:%s\tW_YTD:%lf\n",
		rid_in_warehouses,
		p_warehouse_tmp->W_ID,
		p_warehouse_tmp->W_NAME,	//取名称（W_NAME）
		p_warehouse_tmp->W_STREET_1,	//地址1（W_STREET_1）
		p_warehouse_tmp->W_STREET_2,	//地址2（W_STREET_2）
		p_warehouse_tmp->W_CITY,	//城市（W_CITY）
		p_warehouse_tmp->W_STATE,	//州（W_STATE）
		p_warehouse_tmp->W_ZIP,	//邮政编码（W_ZIP）
		p_warehouse_tmp->W_YTD);	//取累计发生额（W_YTD）
	//void  d_memcpy(void *des, void *src, int size)
	d_memcpy(&warehouse_tmp,p_warehouse_tmp,sizeof(struct warehouse));
	warehouse_tmp.W_YTD+=pay_H_AMOUNT;
	//void update(int table_type, int record_id, void *record);
	update(WAREHOUSE,rid_in_warehouses,&warehouse_tmp);	//累计发生金额加上支付金额
	// printf("rid_in_warehouses:%d\tW_ID:%ld\tW_NAME:%s\tW_STREET_1:%s\tW_STREET_2:%s\tW_CITY:%s\tW_STATE:%s\tW_ZIP:%s\tW_YTD:%lf\n",
	// 	rid_in_warehouses,
	// 	p_warehouse_tmp->W_ID,
	// 	p_warehouse_tmp->W_NAME,	//取名称（W_NAME）
	// 	p_warehouse_tmp->W_STREET_1,	//地址1（W_STREET_1）
	// 	p_warehouse_tmp->W_STREET_2,	//地址2（W_STREET_2）
	// 	p_warehouse_tmp->W_CITY,	//城市（W_CITY）
	// 	p_warehouse_tmp->W_STATE,	//州（W_STATE）
	// 	p_warehouse_tmp->W_ZIP,	//邮政编码（W_ZIP）
	// 	p_warehouse_tmp->W_YTD);	//取累计发生额（W_YTD）
	// //void  d_memcpy(void *des, void *src, int size)
	// d_memcpy(&warehouse_tmp,p_warehouse_tmp,sizeof(struct warehouse));


	//在地区表（districts）中查询地区代码（D_W_ID,D_ID）。取名称（D_NAME）、地址1（D_STREET_1）、地址2（D_STREET_2）、城市（D_CITY）、州（D_STATE）、邮政编码（D_ZIP）、累计发生额（D_YTD）。
	struct district district_tmp;
	struct district *p_district_tmp;
	int rid_in_districts=0;
	while(1){
		rid_in_districts = table_scan(DISTRICT, LONG, sizeof(long int), ((long)&(district_tmp.D_W_ID)-(long)&(district_tmp.D_ID)), EQ, &pay_D_W_ID, rid_in_districts);	//扫描整张districts表找出D_W_ID为pay_D_W_ID的记录号
		if(rid_in_districts == -1){	//warehouses中没有pay_D_W_ID对应的记录 
			printf("Transaction Exception:\tD_W_ID is not in district table!");
			return;
		}
		p_district_tmp=(struct district *)get(DISTRICT, rid_in_districts);	//获取pay_D_W_ID对应的记录
		if(p_district_tmp->D_ID == pay_D_ID){	//district中有pay_D_W_ID,pay_D_ID对应的记录
			break;
		}else{
			rid_in_districts++;
		}
	}
	printf("rid_in_districts:%d\tD_W_ID:%ld\tD_ID:%ld\tD_NAME:%s\tD_STREET_1:%s\tD_STREET_2:%s\tD_CITY:%s\tD_STATE:%s\tD_ZIP:%s\tD_YTD:%lf\n",
		rid_in_districts,
		p_district_tmp->D_W_ID,
		p_district_tmp->D_ID,
		p_district_tmp->D_NAME,	//取名称（D_NAME）
		p_district_tmp->D_STREET_1,	//地址1（D_STREET_1）
		p_district_tmp->D_STREET_2,	//地址2（D_STREET_2）
		p_district_tmp->D_CITY,	//城市（D_CITY）
		p_district_tmp->D_STATE,	//州（D_STATE）
		p_district_tmp->D_ZIP,	//邮政编码（D_ZIP）
		p_district_tmp->D_YTD);	//取累计发生额（D_YTD）
	//void  d_memcpy(void *des, void *src, int size)
	d_memcpy(&district_tmp,p_district_tmp,sizeof(struct district));
	district_tmp.D_YTD+=pay_H_AMOUNT;
	//void update(int table_type, int record_id, void *record);
	update(DISTRICT,rid_in_districts,&district_tmp);	//累计发生金额加上支付金额
	// printf("rid_in_districts:%d\tD_W_ID:%ld\tD_ID:%ld\tD_NAME:%s\tD_STREET_1:%s\tD_STREET_2:%s\tD_CITY:%s\tD_STATE:%s\tD_ZIP:%s\tD_YTD:%lf\n",
	// 	rid_in_districts,
	// 	p_district_tmp->D_W_ID,
	// 	p_district_tmp->D_ID,
	// 	p_district_tmp->D_NAME,	//取名称（D_NAME）
	// 	p_district_tmp->D_STREET_1,	//地址1（D_STREET_1）
	// 	p_district_tmp->D_STREET_2,	//地址2（D_STREET_2）
	// 	p_district_tmp->D_CITY,	//城市（D_CITY）
	// 	p_district_tmp->D_STATE,	//州（D_STATE）
	// 	p_district_tmp->D_ZIP,	//邮政编码（D_ZIP）
	// 	p_district_tmp->D_YTD);	//取累计发生额（D_YTD）


	//在客户表（customers）中查询地区代码（C_W_ID,C_D_ID,C_ID）。
	struct customer customer_tmp;
	struct customer *p_customer_tmp;
	int rid_in_customers=0;
	while(1){
		rid_in_customers = table_scan(CUSTOMER, LONG, sizeof(long int), ((long)&(customer_tmp.C_W_ID)-(long)&(customer_tmp.C_ID)), EQ, &pay_C_W_ID, rid_in_customers);	//扫描整张customer表找出C_W_ID为pay_C_W_ID的记录号
		if(rid_in_customers == -1){	//customers中没有pay_C_W_ID对应的记录 
			printf("Transaction Exception:\tC_W_ID is not in customer table!\n");
			return;
		}
		p_customer_tmp=(struct customer *)get(CUSTOMER, rid_in_customers);	//获取pay_C_W_ID对应的记录
		if(p_customer_tmp->C_D_ID == pay_C_D_ID && p_customer_tmp->C_ID == pay_C_ID){	//customers中有pay_C_W_ID,pay_C_D_ID,pay_C_ID对应的记录
			break;
		}else{
			rid_in_customers++;
		}
	}
	printf("rid_in_customers:%d\tC_W_ID:%ld\tC_D_ID:%ld\tC_ID:%ld\tC_FIRST:%s\tC_MIDDLE:%s\tC_LAST:%s\tC_STREET_1:%s\tC_STREET_2:%s\tC_CITY:%s\tC_STATE:%s\tC_ZIP:%s\tC_PHONE:%s\tC_SINCE:%ld\tC_CREDIT:%s\tC_CREDIT_LIM:%lf\tC_DISCOUNT:%lf\tC_BALANCE:%lf\n",
		rid_in_customers,
		p_customer_tmp->C_W_ID,
		p_customer_tmp->C_D_ID,
		p_customer_tmp->C_ID,
		p_customer_tmp->C_FIRST,	//取姓名
		p_customer_tmp->C_MIDDLE,
		p_customer_tmp->C_LAST,
		p_customer_tmp->C_STREET_1,	//地址1
		p_customer_tmp->C_STREET_2,	//地址2
		p_customer_tmp->C_CITY,	//城市
		p_customer_tmp->C_STATE,	//州
		p_customer_tmp->C_ZIP,	//邮政编码
		p_customer_tmp->C_PHONE,	//电话
		p_customer_tmp->C_SINCE,	//登记日期
		p_customer_tmp->C_CREDIT,	//信用
		p_customer_tmp->C_CREDIT_LIM,	//超支限额
		p_customer_tmp->C_DISCOUNT,	//折扣
		p_customer_tmp->C_BALANCE);	//欠款余额
	//void  d_memcpy(void *des, void *src, int size)
	d_memcpy(&customer_tmp,p_customer_tmp,sizeof(struct customer));
	customer_tmp.C_BALANCE-=pay_H_AMOUNT;	//欠款余额减去支付金额
	customer_tmp.C_YTD_PAYMENT+=pay_H_AMOUNT;	//累计发生金额加上支付金额
	customer_tmp.C_PAYMENT_CNT++;	//支付次数加一
	//void update(int table_type, int record_id, void *record);
	update(CUSTOMER,rid_in_customers,&customer_tmp);	//修改记录
	// printf("rid_in_customers:%d\tC_W_ID:%ld\tC_D_ID:%ld\tC_ID:%ld\tC_FIRST:%s\tC_MIDDLE:%s\tC_LAST:%s\tC_STREET_1:%s\tC_STREET_2:%s\tC_CITY:%s\tC_STATE:%s\tC_ZIP:%s\tC_PHONE:%s\tC_SINCE:%ld\tC_CREDIT:%s\tC_CREDIT_LIM:%lf\tC_DISCOUNT:%lf\tC_BALANCE:%lf\n",
	// 	rid_in_customers,
	// 	p_customer_tmp->C_W_ID,
	// 	p_customer_tmp->C_D_ID,
	// 	p_customer_tmp->C_ID,
	// 	p_customer_tmp->C_FIRST,	//取姓名
	// 	p_customer_tmp->C_MIDDLE,
	// 	p_customer_tmp->C_LAST,
	// 	p_customer_tmp->C_STREET_1,	//地址1
	// 	p_customer_tmp->C_STREET_2,	//地址2
	// 	p_customer_tmp->C_CITY,	//城市
	// 	p_customer_tmp->C_STATE,	//州
	// 	p_customer_tmp->C_ZIP,	//邮政编码
	// 	p_customer_tmp->C_PHONE,	//电话
	// 	p_customer_tmp->C_SINCE,	//登记日期
	// 	p_customer_tmp->C_CREDIT,	//信用
	// 	p_customer_tmp->C_CREDIT_LIM,	//超支限额
	// 	p_customer_tmp->C_DISCOUNT,	//折扣
	// 	p_customer_tmp->C_BALANCE);	//欠款余额


	//如果信用（C_CREDIT）为“BC”,则客户备注(C_DATA)右移，左端插入客户代码（C_W_ID、C_D_ID、C_ID）、地区代码（D_ID）、仓库代码（W_ID）、支付金额（H_AMOUNT）
	if(d_strcmp(customer_tmp.C_CREDIT,"BC")==0){
		char pay_C_DATA[500+1];
		char char_C_W_ID[50];
		char char_C_D_ID[50];
		char char_C_ID[50];
		char char_D_ID[50];
		char char_W_ID[50];
		d_ltoa(customer_tmp.C_W_ID,char_C_W_ID);	//客户代码（C_W_ID、C_D_ID、C_ID）
		d_ltoa(customer_tmp.C_D_ID,char_C_D_ID);
		d_ltoa(customer_tmp.C_ID,char_C_ID);
		d_ltoa(pay_D_ID,char_D_ID);	//地区代码（D_ID）
		d_ltoa(pay_W_ID,char_W_ID);	//仓库代码（W_ID）
		char *separator=";";
		d_str_append(pay_C_DATA,char_C_W_ID,501);
		d_str_append(pay_C_DATA,separator,501);
		d_str_append(pay_C_DATA,char_C_D_ID,501);
		d_str_append(pay_C_DATA,separator,501);
		d_str_append(pay_C_DATA,char_C_ID,501);
		d_str_append(pay_C_DATA,separator,501);
		d_str_append(pay_C_DATA,char_D_ID,501);
		d_str_append(pay_C_DATA,separator,501);
		d_str_append(pay_C_DATA,char_W_ID,501);
		d_str_append(pay_C_DATA,separator,501);
		d_str_append(pay_C_DATA,customer_tmp.C_DATA,501);
		d_memcpy(customer_tmp.C_DATA,pay_C_DATA,501);
		update(CUSTOMER,rid_in_customers,&customer_tmp);	//修改记录
		//printf("C_DATA:%s\n",p_customer_tmp->C_DATA);
	}


	//向历史记录表（historys）中添加一条记录，备注（H_DATA）写为仓库名称（W_NAME）和地区名称（D_NAME）
	struct history history_tmp;
	history_tmp.H_C_ID=pay_C_ID;
	history_tmp.H_C_D_ID=pay_C_D_ID;
	history_tmp.H_C_W_ID=pay_C_W_ID;
	history_tmp.H_D_ID=pay_D_ID;
	history_tmp.H_W_ID=pay_W_ID;
	history_tmp.H_DATE=1184;
	history_tmp.H_AMOUNT=pay_H_AMOUNT;
	int pay_W_NAME_length=d_strlen(p_warehouse_tmp->W_NAME);
	d_memcpy(history_tmp.H_DATA,p_warehouse_tmp->W_NAME,pay_W_NAME_length);
	history_tmp.H_DATA[pay_W_NAME_length]=';';
	d_memcpy(history_tmp.H_DATA+(pay_W_NAME_length+1),p_district_tmp->D_NAME,d_strlen(p_district_tmp->D_NAME));
	//int insert_rec(int table_type, void *record);
	insert_rec(HISTORY,&history_tmp);

	//事务提交 
	printf("Payment Transaction is finished!\n");
}