#include "hip/hip_runtime.h"
//
//  	desc: tpcc benchmark implementation in GPU
//	date: 2018-3-27
//	author: Xie Shangwei
//


#include <stdlib.h>
#include <string.h>
#include <stdio.h>
#include <assert.h>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include "tpcc_table.h"
#include "utility.h"
#include "table_operator.h"
#include "tx.h"

int get_item(struct item *item_arr);
int get_warehouse(struct warehouse *warehouse_arr);
int get_order(struct order *order);
int get_customer(struct customer *c);
int get_new_order(struct new_order *n);
int get_order_line(struct order_line *o);
int get_district(struct district *d);
int get_stock(struct stock *s);
int get_history(struct history *h);

//get 9 tables.
struct warehouse h_warehouses[MAX_WAREHOUSE_NUM];
struct district h_districts[MAX_DISTRICT_NUM];
struct customer h_customers[MAX_CUSTOMER_NUM];
struct history h_historys[MAX_HISTORY_NUM];
struct new_order h_neworders[MAX_NEWORDER_NUM];
struct order h_orders[MAX_ORDER_NUM];
struct order_line h_orderlines[MAX_ORDER_LINE_NUM];
struct stock h_stocks[MAX_STOCK_NUM];
struct item h_items[MAX_ITEM_NUM];

//tables's slot flag, mark the slot is used or not.
char  h_warehouses_flag[MAX_WAREHOUSE_NUM];
char  h_districts_flag[MAX_DISTRICT_NUM];   
char  h_customers_flag[MAX_CUSTOMER_NUM];   
char  h_historys_flag[MAX_HISTORY_NUM];      
char  h_neworders_flag[MAX_NEWORDER_NUM];  
char  h_orders_flag[MAX_ORDER_NUM];            
char  h_orderlines_flag[MAX_ORDER_LINE_NUM];  
char  h_items_flag[MAX_ITEM_NUM];               
char  h_stocks_flag[MAX_STOCK_NUM];

struct warehouse *h_d_warehouses;
struct district *h_d_districts;
struct customer *h_d_customers;
struct history *h_d_historys;
struct new_order *h_d_new_orders;
struct order *h_d_orders;
struct order_line *h_d_orderlines;
struct item *h_d_items;
struct stock *h_d_stocks;

char *h_d_warehouses_flag;
char *h_d_districts_flag;
char *h_d_customers_flag;
char *h_d_historys_flag;
char *h_d_new_orders_flag;
char *h_d_orders_flag;
char *h_d_orderlines_flag;
char *h_d_items_flag;
char *h_d_stocks_flag;

__device__ char *d_warehouses_flag;
__device__ char *d_districts_flag;
__device__ char *d_customers_flag;
__device__ char *d_historys_flag;
__device__ char *d_new_orders_flag;
__device__ char *d_orders_flag;
__device__ char *d_orderlines_flag;
__device__ char *d_items_flag;
__device__ char *d_stocks_flag;

__device__ struct warehouse *d_warehouses;
__device__ struct district *d_districts;
__device__ struct customer *d_customers;
__device__ struct history *d_historys;
__device__ struct new_order *d_new_orders;
__device__ struct order *d_orders;
__device__ struct order_line *d_orderlines;
__device__ struct item *d_items;
__device__ struct stock *d_stocks;

__global__
void cp_table_to_device(
	struct warehouse *h_d_warehouses,
	struct district *h_d_districts,
	struct customer *h_d_customers,
	struct history *h_d_historys,
	struct new_order *h_d_new_orders,
	struct order *h_d_orders,
	struct order_line *h_d_orderlines,
	struct item *h_d_items,
	struct stock *h_d_stocks);

__global__
void cp_flag_to_device(
	char *h_d_warehouses_flag,
	char *h_d_districts_flag,
	char *h_d_customers_flag,
	char *h_d_historys_flag,
	char *h_d_new_orders_flag,
	char *h_d_orders_flag,
	char *h_d_orderlines_flag,
	char *h_d_items_flag,
	char *h_d_stocks_flag);

__global__
void transaction_process(){
	//tx_stock_level();
	tx_new_order();
	tx_payment();
}

void load_data();
void cp_data_to_dev();

int main(int argc, char **argv){
	load_data();

	cp_data_to_dev();

	cp_table_to_device<<<1, 1>>>(
			h_d_warehouses,
			h_d_districts,
			h_d_customers,
			h_d_historys,
			h_d_new_orders,
			h_d_orders,
			h_d_orderlines,
			h_d_items,
			h_d_stocks);

	cp_flag_to_device<<<1, 1>>>(
			h_d_warehouses_flag,
			h_d_districts_flag,
			h_d_customers_flag,
			h_d_historys_flag,
			h_d_new_orders_flag,
			h_d_orders_flag,
			h_d_orderlines_flag,
			h_d_items_flag,
			h_d_stocks_flag);

	transaction_process<<<1, 1>>>();
	
	hipMemcpy(h_warehouses_flag, h_d_warehouses_flag, sizeof(char)*MAX_WAREHOUSE_NUM, hipMemcpyDeviceToHost);
	
	return 0;
}

void load_data(){
		int warehouse_num = get_warehouse(h_warehouses);
		int i;
		for(i = 0; i<warehouse_num ; i++){
			h_warehouses_flag[i] = 1;
		}

		int stock_num = get_stock(h_stocks);
		for( i = 0; i<stock_num ; i++){
			h_stocks_flag[i] = 1;
		}

		int district_num = get_district(h_districts);
		for( i = 0; i<district_num ; i++){
			h_districts_flag[i] = 1;
		}

		int customer_num = get_customer(h_customers);
		for( i = 0; i<customer_num ; i++){
			h_customers_flag[i] = 1;
		}

		int new_order_num = get_new_order(h_neworders);
		for( i = 0; i<new_order_num ; i++){
			h_neworders_flag[i] = 1;
		}

		int order_num = get_order(h_orders);
		for( i = 0; i<order_num ; i++){
			h_orders_flag[i] = 1;
		}

		int order_line_num = get_order_line(h_orderlines);
		for( i = 0; i<order_line_num ; i++){
			h_orderlines_flag[i] = 1;
		}

		int item_num = get_item(h_items);
		for( i = 0; i<item_num ; i++){
			h_items_flag[i] = 1;
		}

		int history_num = get_history(h_historys);
		for( i = 0; i<history_num ; i++){
			h_historys_flag[i] = 1;
		}
		printf("load data succeed!\n");
}

void cp_data_to_dev(){
	hipMalloc( (void **)&h_d_warehouses, sizeof(struct warehouse)*(MAX_WAREHOUSE_NUM));
	hipMalloc( (void **)&h_d_districts,  sizeof(struct district)*(MAX_DISTRICT_NUM));
	hipMalloc( (void **)&h_d_customers,  sizeof(struct customer)*(MAX_CUSTOMER_NUM));
	hipMalloc( (void **)&h_d_historys,  sizeof(struct history)*(MAX_HISTORY_NUM));
	hipMalloc( (void **)&h_d_new_orders,  sizeof(struct new_order)*(MAX_NEWORDER_NUM));
	hipMalloc( (void **)&h_d_orders,  sizeof(struct order)*(MAX_ORDER_NUM));
	hipMalloc( (void **)&h_d_orderlines,  sizeof(struct order_line)*(MAX_ORDER_LINE_NUM));
	hipMalloc( (void **)&h_d_items,  sizeof(struct item)*(MAX_ITEM_NUM));
	hipMalloc( (void **)&h_d_stocks,  sizeof(struct stock)*(MAX_STOCK_NUM));

	hipMalloc( (void **)&h_d_warehouses_flag, sizeof(char)*MAX_WAREHOUSE_NUM);
	hipMalloc( (void **)&h_d_districts_flag, sizeof(char)*MAX_DISTRICT_NUM);
	hipMalloc( (void **)&h_d_customers_flag, sizeof(char)*MAX_CUSTOMER_NUM);
	hipMalloc( (void **)&h_d_historys_flag, sizeof(char)*MAX_HISTORY_NUM);
	hipMalloc( (void **)&h_d_new_orders_flag, sizeof(char)*MAX_NEWORDER_NUM);
	hipMalloc( (void **)&h_d_orders_flag, sizeof(char)*MAX_ORDER_NUM);
	hipMalloc( (void **)&h_d_orderlines_flag, sizeof(char)*MAX_ORDER_LINE_NUM);
	hipMalloc( (void **)&h_d_items_flag, sizeof(char)*MAX_ITEM_NUM);
	hipMalloc( (void **)&h_d_stocks_flag, sizeof(char)*MAX_STOCK_NUM);
	printf("device memory allocate succeed.\n");	

	hipMemcpy(h_d_warehouses, h_warehouses, sizeof(struct warehouse)*(MAX_WAREHOUSE_NUM), hipMemcpyHostToDevice);
	hipMemcpy(h_d_districts, h_districts, sizeof(struct district)*MAX_DISTRICT_NUM, hipMemcpyHostToDevice);
	hipMemcpy(h_d_customers, h_customers, sizeof(struct customer)*MAX_CUSTOMER_NUM, hipMemcpyHostToDevice);
	hipMemcpy(h_d_historys, h_historys, sizeof(struct history)*MAX_HISTORY_NUM, hipMemcpyHostToDevice);
	hipMemcpy(h_d_new_orders, h_neworders, sizeof(struct new_order)*MAX_NEWORDER_NUM, hipMemcpyHostToDevice);
	hipMemcpy(h_d_orders, h_orders, sizeof(struct order)*MAX_ORDER_NUM, hipMemcpyHostToDevice);
	hipMemcpy(h_d_orderlines, h_orderlines, sizeof(struct order_line)*MAX_ORDER_LINE_NUM, hipMemcpyHostToDevice);
	hipMemcpy(h_d_items, h_items, sizeof(struct item)*MAX_ITEM_NUM, hipMemcpyHostToDevice);
	hipMemcpy(h_d_stocks, h_stocks, sizeof(struct stock)*MAX_STOCK_NUM, hipMemcpyHostToDevice);
	
	hipMemcpy(h_d_warehouses_flag, h_warehouses_flag, sizeof(char)*(MAX_WAREHOUSE_NUM), hipMemcpyHostToDevice);
	hipMemcpy(h_d_districts_flag, h_districts_flag, sizeof(char)*MAX_DISTRICT_NUM, hipMemcpyHostToDevice);
	hipMemcpy(h_d_customers_flag, h_customers_flag, sizeof(char)*MAX_CUSTOMER_NUM, hipMemcpyHostToDevice);
	hipMemcpy(h_d_historys_flag, h_historys_flag, sizeof(char)*MAX_HISTORY_NUM, hipMemcpyHostToDevice);
	hipMemcpy(h_d_new_orders_flag, h_neworders_flag, sizeof(char)*MAX_NEWORDER_NUM, hipMemcpyHostToDevice);
	hipMemcpy(h_d_orders_flag, h_orders_flag, sizeof(char)*MAX_ORDER_NUM, hipMemcpyHostToDevice);
	hipMemcpy(h_d_orderlines_flag, h_orderlines_flag, sizeof(char)*MAX_ORDER_LINE_NUM, hipMemcpyHostToDevice);
	hipMemcpy(h_d_items_flag, h_items_flag, sizeof(char)*MAX_ITEM_NUM, hipMemcpyHostToDevice);
	hipMemcpy(h_d_stocks_flag, h_stocks_flag, sizeof(char)*MAX_STOCK_NUM, hipMemcpyHostToDevice);
	printf("memcpy succeed.\n");
}

__global__
void cp_table_to_device(
	struct warehouse *h_d_warehouses,
	struct district *h_d_districts,
	struct customer *h_d_customers,
	struct history *h_d_historys,
	struct new_order *h_d_new_orders,
	struct order *h_d_orders,
	struct order_line *h_d_orderlines,
	struct item *h_d_items,
	struct stock *h_d_stocks){

	d_warehouses = h_d_warehouses;
	d_districts = h_d_districts;
	d_customers = h_d_customers;
	d_historys = h_d_historys;
	d_new_orders = h_d_new_orders;
	d_orders = h_d_orders;
	d_orderlines = h_d_orderlines;
	d_items = h_d_items;
	d_stocks = h_d_stocks;
}

__global__
void cp_flag_to_device(
	char *h_d_warehouses_flag,
	char *h_d_districts_flag,
	char *h_d_customers_flag,
	char *h_d_historys_flag,
	char *h_d_new_orders_flag,
	char *h_d_orders_flag,
	char *h_d_orderlines_flag,
	char *h_d_items_flag,
	char *h_d_stocks_flag){

	d_warehouses_flag = h_d_warehouses_flag;
	d_districts_flag = h_d_districts_flag;
	d_customers_flag = h_d_customers_flag;
	d_historys_flag = h_d_historys_flag;
	d_new_orders_flag = h_d_new_orders_flag;
	d_orders_flag = h_d_orders_flag;
	d_orderlines_flag = h_d_orderlines_flag;
	d_items_flag = h_d_items_flag;
	d_stocks_flag = h_d_stocks_flag;
}
