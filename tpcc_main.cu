#include "hip/hip_runtime.h"
//
//  	desc: tpcc benchmark implementation in GPU
//	date: 2018-3-27
//	author: Xie Shangwei
//


#include <stdlib.h>
#include <string.h>
#include <stdio.h>
#include <assert.h>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include "tpcc_table.h"
#include "utility.h"
#include "table_operator.h"

int get_item(struct item *item_arr);
int get_warehouse(struct warehouse *warehouse_arr);
int get_order(struct order *order);
int get_customer(struct customer *c);
int get_new_order(struct new_order *n);
int get_order_line(struct order_line *o);
int get_district(struct district *d);
int get_stock(struct stock *s);
int get_history(struct history *h);

//get 9 tables.
struct warehouse h_warehouses[MAX_WAREHOUSE_NUM];
struct district h_districts[MAX_DISTRICT_NUM];
struct customer h_customers[MAX_CUSTOMER_NUM];
struct history h_historys[MAX_HISTORY_NUM];
struct new_order h_neworders[MAX_NEWORDER_NUM];
struct order h_orders[MAX_ORDER_NUM];
struct order_line h_orderlines[MAX_ORDER_LINE_NUM];
struct stock h_stocks[MAX_STOCK_NUM];
struct item h_items[MAX_ITEM_NUM];

//tables's slot flag, mark the slot is used or not.
char  h_warehouses_flag[MAX_WAREHOUSE_NUM];
char  h_districts_flag[MAX_DISTRICT_NUM];   
char  h_customers_flag[MAX_CUSTOMER_NUM];   
char  h_historys_flag[MAX_HISTORY_NUM];      
char  h_neworders_flag[MAX_NEWORDER_NUM];  
char  h_orders_flag[MAX_ORDER_NUM];            
char  h_orderlines_flag[MAX_ORDER_LINE_NUM];  
char  h_items_flag[MAX_ITEM_NUM];               
char  h_stocks_flag[MAX_STOCK_NUM];

struct warehouse *h_d_warehouses;
struct district *h_d_districts;
struct customer *h_d_customers;
struct history *h_d_historys;
struct new_order *h_d_new_orders;
struct order *h_d_orders;
struct order_line *h_d_orderlines;
struct item *h_d_items;
struct stock *h_d_stocks;

char *h_d_warehouses_flag;
char *h_d_districts_flag;
char *h_d_customers_flag;
char *h_d_historys_flag;
char *h_d_new_orders_flag;
char *h_d_orders_flag;
char *h_d_orderlines_flag;
char *h_d_items_flag;
char *h_d_stocks_flag;

__device__ char *d_warehouses_flag;
__device__ char *d_districts_flag;
__device__ char *d_customers_flag;
__device__ char *d_historys_flag;
__device__ char *d_new_orders_flag;
__device__ char *d_orders_flag;
__device__ char *d_orderlines_flag;
__device__ char *d_items_flag;
__device__ char *d_stocks_flag;

__device__ struct warehouse *d_warehouses;
__device__ struct district *d_districts;
__device__ struct customer *d_customers;
__device__ struct history *d_historys;
__device__ struct new_order *d_new_orders;
__device__ struct order *d_orders;
__device__ struct order_line *d_orderlines;
__device__ struct item *d_items;
__device__ struct stock *d_stocks;


__global__
void test_table_scan(struct district *h_d_districts, char *h_d_districts_flag){
	printf("into test tabel scan!\n");
	d_districts = h_d_districts;
	d_districts_flag = h_d_districts_flag;
	
	int rid = 0;
	struct district tmp;
	long offset = (unsigned int)&tmp.D_ZIP - (unsigned int)&tmp.D_ID;


	struct district *dist_tmp;
	char tmp_zip[10]="292511111";
	printf("%s\n", tmp_zip);

	rid = table_scan(DISTRICT, STR, 10-1, offset, EQ, tmp_zip, rid);
	while(rid != -1){
		printf("rid\t%d\t:", rid);
		dist_tmp =(struct district *)get(DISTRICT, rid);
		//d_memcpy((void *)&ware_tmp, result, sizeof(struct warehouse));
		
		//test!!!!!!!!!!!!!
		printf("%ld, %ld, %s, %s, %s, %s, %s, %s, %lf, %lf, %ld\n",dist_tmp->D_ID, dist_tmp->D_W_ID,
	  		dist_tmp->D_NAME, dist_tmp->D_STREET_1, dist_tmp->D_STREET_2, dist_tmp->D_CITY, dist_tmp->D_STATE,
	  		dist_tmp->D_ZIP, dist_tmp->D_TAX, dist_tmp->D_YTD, dist_tmp->D_NEXT_O_ID);
		
		rid = table_scan(DISTRICT, STR, 10-1, offset, EQ, tmp_zip, rid+1);
	}
	printf("finish test tabel scan!\n");
	
}

//void insert_rec(int table_type, void *record)
__global__
void test_insert(struct warehouse *h_d_warehouses, char *h_d_warehouses_flag){
	printf("into test insert!\n");
	d_warehouses = h_d_warehouses;
	d_warehouses_flag = h_d_warehouses_flag;
	
	struct warehouse new_rec={5,"huangxiang","asdasdas","asdas","hefei","sd","7854sda",25.3,10.56};
	insert_rec(WAREHOUSE,&new_rec);
	printf("finish test insert!\n");
}

//void update(int table_type, int record_id, void *record);
__global__
void test_update(struct warehouse *h_d_warehouses, char *h_d_warehouses_flag){
	printf("into test update!\n");
	d_warehouses = h_d_warehouses;
	d_warehouses_flag = h_d_warehouses_flag;
	
	struct warehouse update_rec={4,"xsw","asdasdas","asdas","hefei","sd","7854sda",25.3,10.56};
	update(WAREHOUSE,1,&update_rec);
	printf("finish test update!\n");
}

//void delete_rec(int table_type, int record_id);
__global__
void test_delete(struct warehouse *h_d_warehouses, char *h_d_warehouses_flag){
	printf("into test delete!\n");
	d_warehouses = h_d_warehouses;
	d_warehouses_flag = h_d_warehouses_flag;
	
	delete_rec(WAREHOUSE,1);
	delete_rec(WAREHOUSE,3);
	printf("finish test delete!\n");
}


void cp_data_to_dev();

__global__
void cp_table_to_device(
	struct warehouse *h_d_warehouses,
	struct district *h_d_districts,
	struct customer *h_d_customers,
	struct order *h_d_orders,
	struct new_order *h_d_new_orders,
	struct order_line *h_d_orderlines,
	struct item *h_d_items,
	struct stock *h_d_stocks,
	struct history *h_d_historys);

__global__
void cp_flag_to_device(
	char *h_d_warehouses_flag,
	char *h_d_districts_flag,
	char *h_d_orders_flag,
	char *h_d_orderlines_flag,
	char *h_d_new_orders,
	char *h_d_orders,
	char *h_d_customers_flag,
	char *h_d_stocks_flag,
	char *h_d_historys_flag);

__device__
void stock_level(){
	printf("into stock_level\n");
	// random function
	hiprandState state;
	int id = threadIdx.x;
	long rand = 0;
	long seed = rand;
	hiprand_init(seed, id, 0, &state);
	// for(int i = 0; i< 20 ; i++)
	// {
	// 	printf("rand : %u\n", hiprand(&state));
	// }
	
	//generate parameters.
	long W_ID = (long)(hiprand(&state)%3 + 1);
	long D_W_ID;
	long D_ID;
	unsigned int limit;

	int rid = (int)hiprand(&state)%30;
	printf("rid = %d\n", rid);
	rid = table_scan(DISTRICT, LONG, 0, 0, NO, NULL, rid);
	printf("ok\n");
	struct district tmp_district;
	void *content = get(DISTRICT, rid);
	d_memcpy(&tmp_district, content, sizeof(struct district));
	D_W_ID = tmp_district.D_W_ID;
	D_ID = tmp_district.D_ID;
	limit = hiprand(&state);

	printf("****** stock_level ******\nparameters:\n W_ID : %ld\n D_W_ID : %ld\n D_ID : %ld\n limit : %u\n", W_ID, D_W_ID, D_ID, limit);

	int offset_D_W_ID = (unsigned int)&tmp_district.D_W_ID - (unsigned int)&tmp_district.D_ID;
	int offset_D_ID = 0;
	int rid1 = table_scan(DISTRICT, LONG, 0, offset_D_W_ID, EQ, &W_ID, 0);
	int rid2;
	while( rid1 != -1){
		rid2 = table_scan(DISTRICT, LONG, 0, offset_D_ID, EQ, &D_ID, rid);
		if(rid1 == rid2)
			break;
		rid1 = table_scan(DISTRICT, LONG, 0, offset_D_W_ID, EQ, &W_ID, rid1+1);
	}

	content = get(DISTRICT, rid1);
	d_memcpy(&tmp_district, content, sizeof(struct district));

	long next_order_id = tmp_district.D_NEXT_O_ID;
	printf("D_NEXT_ID : %ld\n", next_order_id);


}


__global__
void transaction_process(){
	printf("into kernel !\n");
	
	stock_level();
}

void load_data();


int main(int argc, char **argv){
	load_data();
	printf("load data succeed!\n");

	// test
	// test_table_scan<<<1, 1>>>(h_d_warehouses, h_d_warehouses_flag);
	// test_insert<<<1, 1>>>(h_d_warehouses, h_d_warehouses_flag);
	// test_table_scan<<<1, 1>>>(h_d_warehouses, h_d_warehouses_flag);
	// test_update<<<1, 1>>>(h_d_warehouses, h_d_warehouses_flag);
	// test_table_scan<<<1, 1>>>(h_d_warehouses, h_d_warehouses_flag);
	// test_delete<<<1, 1>>>(h_d_warehouses, h_d_warehouses_flag);
	// test_table_scan<<<1, 1>>>(h_d_warehouses, h_d_warehouses_flag);
 
	
	//test_table_scan<<<1, 1>>>(h_d_districts, h_d_districts_flag);
	cp_data_to_dev();

	cp_table_to_device<<<1, 1>>>(
			h_d_warehouses,
			h_d_districts,
			h_d_customers,
			h_d_orders,
			h_d_new_orders,
			h_d_orderlines,
			h_d_items,
			h_d_stocks,
			h_d_historys);

	cp_flag_to_device<<<1, 1>>>(
			h_d_warehouses_flag,
			h_d_districts_flag,
			h_d_orders_flag,
			h_d_orderlines_flag,
			h_d_new_orders_flag,
			h_d_items_flag,
			h_d_customers_flag,
			h_d_stocks_flag,
			h_d_historys_flag);

	
	transaction_process<<<1, 1>>>();
	
	hipMemcpy(h_warehouses_flag, h_d_warehouses_flag, sizeof(char)*MAX_WAREHOUSE_NUM, hipMemcpyDeviceToHost);
	
	return 0;
}

//void cp_data_to_dev(){
	
//}

void load_data(){
		int warehouse_num = get_warehouse(h_warehouses);
		int i;
		for(i = 0; i<warehouse_num ; i++){
			h_warehouses_flag[i] = (char)1;
			printf("load: h_warehouses_flag %d , val : %d\n", i, h_warehouses_flag[i]);
		}

			

//		printf("load warehouse succeed.\n");		


		int stock_num = get_stock(h_stocks);
		for( i = 0; i<stock_num ; i++){
			h_stocks_flag[i] = 1;
		}


//		printf("load stock succeed.\n");		

		int district_num = get_district(h_districts);
		for( i = 0; i<district_num ; i++){
			h_districts_flag[i] = 1;
		}


//		printf("load district succeed.\n");		
		
		int customer_num = get_customer(h_customers);
		for( i = 0; i<customer_num ; i++){
			h_customers_flag[i] = 1;
		}
//		printf("load customer succeed.\n");		

			
		int new_order_num = get_new_order(h_neworders);
		for( i = 0; i<new_order_num ; i++){
			h_neworders_flag[i] = 1;
		}
//		printf("load new_order succeed.\n");	


		
		int order_num = get_order(h_orders);
		for( i = 0; i<order_num ; i++){
			h_orders_flag[i] = 1;
		}
//		printf("load order_num succeed.\n");		


	
		int order_line_num = get_order_line(h_orderlines);
		for( i = 0; i<order_line_num ; i++){
			h_orderlines_flag[i] = 1;
		}
//		printf("load order_line succeed.\n");	


		
		int item_num = get_item(h_items);
		for( i = 0; i<item_num ; i++){
			h_items_flag[i] = 1;
		}		
//		printf("load item succeed.\n");	
		int history_num = get_history(h_historys);
		for( i = 0; i<history_num ; i++){
			h_historys_flag[i] = 1;
		}
//		printf("load history succeed.\n");		

		
}

void cp_data_to_dev(){
	hipMalloc( (void **)&h_d_warehouses, sizeof(struct warehouse)*(MAX_WAREHOUSE_NUM));
	hipMalloc( (void **)&h_d_districts,  sizeof(struct district)*(MAX_DISTRICT_NUM));
	hipMalloc( (void **)&h_d_customers,  sizeof(struct customer)*(MAX_CUSTOMER_NUM));
	hipMalloc( (void **)&h_d_historys,  sizeof(struct history)*(MAX_HISTORY_NUM));
	hipMalloc( (void **)&h_d_new_orders,  sizeof(struct new_order)*(MAX_NEWORDER_NUM));
	hipMalloc( (void **)&h_d_orders,  sizeof(struct order)*(MAX_ORDER_NUM));
	hipMalloc( (void **)&h_d_orderlines,  sizeof(struct order_line)*(MAX_ORDER_LINE_NUM));
	hipMalloc( (void **)&h_d_items,  sizeof(struct item)*(MAX_ITEM_NUM));
	hipMalloc( (void **)&h_d_stocks,  sizeof(struct stock)*(MAX_STOCK_NUM));

	hipMalloc( (void **)&h_d_warehouses_flag, sizeof(char)*MAX_WAREHOUSE_NUM);
	hipMalloc( (void **)&h_d_districts_flag, sizeof(char)*MAX_DISTRICT_NUM);
	hipMalloc( (void **)&h_d_customers_flag, sizeof(char)*MAX_CUSTOMER_NUM);
	hipMalloc( (void **)&h_d_historys_flag, sizeof(char)*MAX_HISTORY_NUM);
	hipMalloc( (void **)&h_d_new_orders_flag, sizeof(char)*MAX_NEWORDER_NUM);
	hipMalloc( (void **)&h_d_orders_flag, sizeof(char)*MAX_ORDER_NUM);
	hipMalloc( (void **)&h_d_orderlines_flag, sizeof(char)*MAX_ORDER_LINE_NUM);
	hipMalloc( (void **)&h_d_items_flag, sizeof(char)*MAX_ITEM_NUM);
	hipMalloc( (void **)&h_d_stocks_flag, sizeof(char)*MAX_STOCK_NUM);
	printf("device memory allocate succeed.\n");	

	hipMemcpy(h_d_warehouses, h_warehouses, sizeof(struct warehouse)*(MAX_WAREHOUSE_NUM), hipMemcpyHostToDevice);
	hipMemcpy(h_d_districts, h_districts, sizeof(struct district)*MAX_DISTRICT_NUM, hipMemcpyHostToDevice);
	hipMemcpy(h_d_customers, h_customers, sizeof(struct customer)*MAX_CUSTOMER_NUM, hipMemcpyHostToDevice);
	hipMemcpy(h_d_historys, h_historys, sizeof(struct history)*MAX_HISTORY_NUM, hipMemcpyHostToDevice);
	hipMemcpy(h_d_new_orders, h_neworders, sizeof(struct new_order)*MAX_NEWORDER_NUM, hipMemcpyHostToDevice);
	hipMemcpy(h_d_orders, h_orders, sizeof(struct order)*MAX_ORDER_NUM, hipMemcpyHostToDevice);
	hipMemcpy(h_d_orderlines, h_orderlines, sizeof(struct order_line)*MAX_ORDER_LINE_NUM, hipMemcpyHostToDevice);
	hipMemcpy(h_d_items, h_items, sizeof(struct item)*MAX_ITEM_NUM, hipMemcpyHostToDevice);
	hipMemcpy(h_d_stocks, h_stocks, sizeof(struct stock)*MAX_STOCK_NUM, hipMemcpyHostToDevice);
	
	hipMemcpy(h_d_warehouses_flag, h_warehouses_flag, sizeof(char)*(MAX_WAREHOUSE_NUM), hipMemcpyHostToDevice);
	hipMemcpy(h_d_districts_flag, h_districts_flag, sizeof(char)*MAX_DISTRICT_NUM, hipMemcpyHostToDevice);
	hipMemcpy(h_d_customers_flag, h_customers_flag, sizeof(char)*MAX_CUSTOMER_NUM, hipMemcpyHostToDevice);
	hipMemcpy(h_d_historys_flag, h_historys_flag, sizeof(char)*MAX_HISTORY_NUM, hipMemcpyHostToDevice);
	hipMemcpy(h_d_new_orders_flag, h_neworders_flag, sizeof(char)*MAX_NEWORDER_NUM, hipMemcpyHostToDevice);
	hipMemcpy(h_d_orders_flag, h_orders_flag, sizeof(char)*MAX_ORDER_NUM, hipMemcpyHostToDevice);
	hipMemcpy(h_d_orderlines_flag, h_orderlines_flag, sizeof(char)*MAX_ORDER_LINE_NUM, hipMemcpyHostToDevice);
	hipMemcpy(h_d_items_flag, h_items_flag, sizeof(char)*MAX_ITEM_NUM, hipMemcpyHostToDevice);
	hipMemcpy(h_d_stocks_flag, h_stocks_flag, sizeof(char)*MAX_STOCK_NUM, hipMemcpyHostToDevice);
	printf("memcpy succeed.\n");
}

__global__
void cp_table_to_device(
	struct warehouse *h_d_warehouses,
	struct district *h_d_districts,
	struct customer *h_d_customers,
	struct order *h_d_orders,
	struct new_order *h_d_new_orders,
	struct order_line *h_d_orderlines,
	struct item *h_d_items,
	struct stock *h_d_stocks,
	struct history *h_d_historys){
	printf("cp tabel to device\n");
	d_warehouses = h_d_warehouses;
	d_districts = h_d_districts;
	d_customers = h_d_customers;
	d_orders = h_d_orders;
	d_new_orders = h_d_new_orders;
	d_orderlines = h_d_orderlines;
	d_items = h_d_items;
	d_stocks = h_d_stocks;
	d_historys = h_d_historys;
}

__global__
void cp_flag_to_device(
	char *h_d_warehouses_flag,
	char *h_d_districts_flag,
	char *h_d_orders_flag,
	char *h_d_orderlines_flag,
	char *h_d_new_orders_flag,
	char *h_d_items_flag,
	char *h_d_customers_flag,
	char *h_d_stocks_flag,
	char *h_d_historys_flag){
	printf("cp flag to device\n");
	    	d_warehouses_flag =   h_d_warehouses_flag;
	    	d_districts_flag = h_d_districts_flag;
	    	d_orders_flag = h_d_orders_flag;
	    	d_orderlines_flag = h_d_orderlines_flag;
	    	d_new_orders_flag = h_d_new_orders_flag;
	    	d_items_flag = h_d_items_flag;
	    	d_customers_flag = h_d_customers_flag;
	    	d_stocks_flag = h_d_stocks_flag;
	    	d_historys_flag = h_d_historys_flag;
}
