//
//  	desc: tpcc benchmark implementation in GPU
//	date: 2018-3-27
//	author: Xie Shangwei
//


#include <stdlib.h>
#include <string.h>
#include <stdio.h>

#include <hip/hip_runtime.h>

#include "tpcc_table.h"

void load_items();
void load_warehouses();
void load_orders();
void load_customers();

//get 9 tables.
struct warehouse h_warehouses[MAX_WAREHOUSE_NUM];
struct district h_districts[MAX_DISTRICT_NUM];
struct customer h_customers[MAX_CUSTOMER_NUM];
struct history h_hIstorys[MAX_HISTORY_NUM];
struct new_order h_neworders[MAX_NEWORDER_NUM];
struct order h_orders[MAX_ORDER_NUM];
struct order_line h_orderlines[MAX_ORDER_LINE];
struct item h_items[MAX_ITEM_NUM];
struct stoc h_stocks[MAX_STOCK_NUM];

//tables's slot flag, mark the slot is used or not.
char warehouse h_warehouses_flag[MAX_WAREHOUSE_NUM];
char district h_districts_flag[MAX_DISTRICT_NUM];   
char customer h_customers_flag[MAX_CUSTOMER_NUM];   
char history h_hIstorys_flag[MAX_HISTORY_NUM];      
char new_order h_neworders_flag[MAX_NEWORDER_NUM];  
char order h_orders_flag[MAX_ORDER_NUM];            
char order_line h_orderlines_flag[MAX_ORDER_LINE];  
char item h_items_flag[MAX_ITEM_NUM];               


int main(int argc, char **argv){
		
}


