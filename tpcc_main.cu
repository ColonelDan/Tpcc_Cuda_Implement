#include "hip/hip_runtime.h"
//
//  	desc: tpcc benchmark implementation in GPU
//	date: 2018-3-27
//	author: Xie Shangwei
//


#include <stdlib.h>
#include <string.h>
#include <stdio.h>
#include <assert.h>

#include <hip/hip_runtime.h>

#include "tpcc_table.h"

int get_item(struct item *item_arr);
int get_warehouse(struct warehouse *warehouse_arr);
int get_order(struct order *order);
int get_customer(struct customer *c);
int get_new_order(struct new_order *n);
int get_order_line(struct order_line *o);
int get_district(struct district *d);
int get_stock(struct stock *s);
int get_history(struct history *h);


//get 9 tables.
struct warehouse h_warehouses[MAX_WAREHOUSE_NUM];
struct district h_districts[MAX_DISTRICT_NUM];
struct customer h_customers[MAX_CUSTOMER_NUM];
struct history h_historys[MAX_HISTORY_NUM];
struct new_order h_neworders[MAX_NEWORDER_NUM];
struct order h_orders[MAX_ORDER_NUM];
struct order_line h_orderlines[MAX_ORDER_LINE_NUM];
struct stock h_stocks[MAX_STOCK_NUM];
struct item h_items[MAX_ITEM_NUM];

//tables's slot flag, mark the slot is used or not.
char  h_warehouses_flag[MAX_WAREHOUSE_NUM];
char  h_districts_flag[MAX_DISTRICT_NUM];   
char  h_customers_flag[MAX_CUSTOMER_NUM];   
char  h_historys_flag[MAX_HISTORY_NUM];      
char  h_neworders_flag[MAX_NEWORDER_NUM];  
char  h_orders_flag[MAX_ORDER_NUM];            
char  h_orderlines_flag[MAX_ORDER_LINE_NUM];  
char  h_items_flag[MAX_ITEM_NUM];               
char  h_stocks_flag[MAX_STOCK_NUM];

struct warehouse *h_d_warehouses;
struct district *h_d_districts;
struct customer *h_d_customers;
struct history *h_d_historys;
struct new_order *h_d_new_orders;
struct order *h_d_orders;
struct orderline *h_d_orderlines;
struct item *h_d_items;
struct stock *h_d_stocks;

char *h_d_warehouses_flag;
char *h_d_districts_flag;
char *h_d_customers_flag;
char *h_d_historys_flag;
char *h_d_new_orders_flag;
char *h_d_orders_flag;
char *h_d_orderlines_flag;
char *h_d_items_flag;
char *h_d_stocks_flag;

__device__ char *d_warehouses_flag;
__device__ char *d_districts_flag;
__device__ char *d_customers_flag;
__device__ char *d_historys_flag;
__device__ char *d_new_orders_flag;
__device__ char *d_orders_flag;
__device__ char *d_orderlines_flag;
__device__ char *d_items_flag;
__device__ char *d_stocks_flag;

__device__ struct warehouse *d_warehouses;
__device__ struct district *d_districts;
__device__ struct customer *d_customers;
__device__ struct history *d_historys;
__device__ struct new_order *d_new_orders;
__device__ struct order *d_orders;
__device__ struct orderline *d_orderlines;
__device__ struct item *d_items;
__device__ struct stock *d_stocks;

__device__ 
void d_memcpy(void *des, void *src, int size);

__device__
int d_strcmp(char *des, char *src);

__device__
void insert_rec(int table_type, void *record);

__device__
void delete_rec(int table_type, int record_id);

__device__
void update(int table_type, void *record);

__device__
void *get(int table_type, int rid);

__device__
void get_table_head(int table_type, void **table_head);

__device__
void get_flag_head(int table_type, char **flag_head);

__device__
void get_table_size(int table_type, int *table_size);

__device__
void get_record_size(int table_type, int *record_size);

__device__
int table_scan(int table_type, int attr_type, int attr_size,  int attr_offset, int op , void *value, int rid);

//
// desc : get a free slot id from the table.
// intput : slot flag arry , table type.
// output : the slot id or -1 if there is no
//          free slot exists.
//
__device__
int get_free_slot(char *slot_flag_arry, int table_type);

//
// desc : mark the slot as used by slot id.
//
__device__
void mark_slot_used(char *slot_flag_array, int slot_id);

//
// desc : mark the slot as freed by slot id.
//
__device__
void mark_slot_free(char *slot_flag_array, int slot_id);

__global__
void test_table_scan(struct warehouse *h_d_warehouses, char *h_d_warehouses_flag){
	printf("into kernel\n");
	d_warehouses = h_d_warehouses;
	d_warehouses_flag = h_d_warehouses_flag;
	
	int rid = 0;
	struct warehouse ware_tmp;
	rid = table_scan(WAREHOUSE, LONG, 0, 0, NO, NULL, rid);
	while(rid != -1){
		printf("rid: %d\n", rid);
		void *result = get(WAREHOUSE, rid);
		d_memcpy((void *)&ware_tmp, result, sizeof(struct warehouse));
		printf("W_ID : %d\n", ware_tmp.W_ID);
		rid = table_scan(WAREHOUSE, LONG, 0, 0, NO, NULL, rid+1);
	}
	
}

void load_data();
//void cp_data_to_dev();
int main(int argc, char **argv){
	load_data();
	printf("load data succeed!\n");

	
// memory allocation.
{
	hipMalloc( (void **)&h_d_warehouses, sizeof(struct warehouse)*(MAX_WAREHOUSE_NUM));
	hipMalloc( (void **)&h_d_districts,  sizeof(struct district)*(MAX_DISTRICT_NUM));
	hipMalloc( (void **)&h_d_customers,  sizeof(struct customer)*(MAX_CUSTOMER_NUM));
	hipMalloc( (void **)&h_d_historys,  sizeof(struct history)*(MAX_HISTORY_NUM));
	hipMalloc( (void **)&h_d_new_orders,  sizeof(struct new_order)*(MAX_NEWORDER_NUM));
	hipMalloc( (void **)&h_d_orders,  sizeof(struct order)*(MAX_ORDER_NUM));
	hipMalloc( (void **)&h_d_orderlines,  sizeof(struct order_line)*(MAX_ORDER_LINE_NUM));
	hipMalloc( (void **)&h_d_items,  sizeof(struct item)*(MAX_ITEM_NUM));
	hipMalloc( (void **)&h_d_stocks,  sizeof(struct stock)*(MAX_STOCK_NUM));

	hipMalloc( (void **)&h_d_warehouses_flag, sizeof(char)*MAX_WAREHOUSE_NUM);
	hipMalloc( (void **)&h_d_districts_flag, sizeof(char)*MAX_DISTRICT_NUM);
	hipMalloc( (void **)&h_d_customers_flag, sizeof(char)*MAX_CUSTOMER_NUM);
	hipMalloc( (void **)&h_d_historys_flag, sizeof(char)*MAX_HISTORY_NUM);
	hipMalloc( (void **)&h_d_new_orders_flag, sizeof(char)*MAX_NEWORDER_NUM);
	hipMalloc( (void **)&h_d_orders_flag, sizeof(char)*MAX_ORDER_NUM);
	hipMalloc( (void **)&h_d_orderlines_flag, sizeof(char)*MAX_ORDER_LINE_NUM);
	hipMalloc( (void **)&h_d_items_flag, sizeof(char)*MAX_ITEM_NUM);
	hipMalloc( (void **)&h_d_stocks_flag, sizeof(char)*MAX_STOCK_NUM);
	printf("device memory allocate succeed.\n");	

	hipMemcpy(h_d_warehouses, h_warehouses, sizeof(struct warehouse)*(MAX_WAREHOUSE_NUM), hipMemcpyHostToDevice);
	hipMemcpy(h_d_districts, h_districts, sizeof(struct district)*MAX_DISTRICT_NUM, hipMemcpyHostToDevice);
	hipMemcpy(h_d_customers, h_customers, sizeof(struct customer)*MAX_CUSTOMER_NUM, hipMemcpyHostToDevice);
	hipMemcpy(h_d_historys, h_historys, sizeof(struct history)*MAX_HISTORY_NUM, hipMemcpyHostToDevice);
	hipMemcpy(h_d_new_orders, h_neworders, sizeof(struct new_order)*MAX_NEWORDER_NUM, hipMemcpyHostToDevice);
	hipMemcpy(h_d_orders, h_orders, sizeof(struct order)*MAX_ORDER_NUM, hipMemcpyHostToDevice);
	hipMemcpy(h_d_orderlines, h_orderlines, sizeof(struct order_line)*MAX_ORDER_LINE_NUM, hipMemcpyHostToDevice);
	hipMemcpy(h_d_items, h_items, sizeof(struct item)*MAX_ITEM_NUM, hipMemcpyHostToDevice);
	hipMemcpy(h_d_stocks, h_stocks, sizeof(struct stock)*MAX_STOCK_NUM, hipMemcpyHostToDevice);
	
	hipMemcpy(h_d_warehouses_flag, h_warehouses_flag, sizeof(char)*(MAX_WAREHOUSE_NUM), hipMemcpyHostToDevice);
	hipMemcpy(h_d_districts_flag, h_districts_flag, sizeof(char)*MAX_DISTRICT_NUM, hipMemcpyHostToDevice);
	hipMemcpy(h_d_customers_flag, h_customers_flag, sizeof(char)*MAX_CUSTOMER_NUM, hipMemcpyHostToDevice);
	hipMemcpy(h_d_historys_flag, h_historys_flag, sizeof(char)*MAX_HISTORY_NUM, hipMemcpyHostToDevice);
	hipMemcpy(h_d_new_orders_flag, h_neworders_flag, sizeof(char)*MAX_NEWORDER_NUM, hipMemcpyHostToDevice);
	hipMemcpy(h_d_orders_flag, h_orders_flag, sizeof(char)*MAX_ORDER_NUM, hipMemcpyHostToDevice);
	hipMemcpy(h_d_orderlines_flag, h_orderlines_flag, sizeof(char)*MAX_ORDER_LINE_NUM, hipMemcpyHostToDevice);
	hipMemcpy(h_d_items_flag, h_items_flag, sizeof(char)*MAX_ITEM_NUM, hipMemcpyHostToDevice);
	hipMemcpy(h_d_stocks_flag, h_stocks_flag, sizeof(char)*MAX_STOCK_NUM, hipMemcpyHostToDevice);
	printf("memcpy succeed.\n");
}
	// test table scan.
	test_table_scan<<<1, 1>>>(h_d_warehouses, h_d_warehouses_flag);
	hipMemcpy(h_warehouses_flag, h_d_warehouses_flag, sizeof(char)*MAX_WAREHOUSE_NUM, hipMemcpyDeviceToHost);
	
	return 0;
}

//void cp_data_to_dev(){
	
//}

void load_data(){
		int warehouse_num = get_warehouse(h_warehouses);
		int i;
		for(i = 0; i<warehouse_num ; i++){
			h_warehouses_flag[i] = (char)1;
			printf("load: h_warehouses_flag %d , val : %d\n", i, h_warehouses_flag[i]);
		}

			

//		printf("load warehouse succeed.\n");		


		int stock_num = get_stock(h_stocks);
		for( i = 0; i<stock_num ; i++){
			h_stocks_flag[i] = 1;
		}


//		printf("load stock succeed.\n");		

		int district_num = get_district(h_districts);
		for( i = 0; i<district_num ; i++){
			h_districts_flag[i] = 1;
		}


//		printf("load district succeed.\n");		
		
		int customer_num = get_customer(h_customers);
		for( i = 0; i<customer_num ; i++){
			h_customers_flag[i] = 1;
		}
//		printf("load customer succeed.\n");		

			
		int new_order_num = get_new_order(h_neworders);
		for( i = 0; i<new_order_num ; i++){
			h_neworders_flag[i] = 1;
		}
//		printf("load new_order succeed.\n");	


		
		int order_num = get_order(h_orders);
		for( i = 0; i<order_num ; i++){
			h_orders_flag[i] = 1;
		}
//		printf("load order_num succeed.\n");		


	
		int order_line_num = get_order_line(h_orderlines);
		for( i = 0; i<order_line_num ; i++){
			h_orderlines_flag[i] = 1;
		}
//		printf("load order_line succeed.\n");	


		
		int item_num = get_item(h_items);
		for( i = 0; i<item_num ; i++){
			h_items_flag[i] = 1;
		}		
//		printf("load item succeed.\n");	
		int history_num = get_history(h_historys);
		for( i = 0; i<history_num ; i++){
			h_historys_flag[i] = 1;
		}
//		printf("load history succeed.\n");		

		
}

__device__
void delete_rec(int table_type, int record_id){
	const int bid = blockIdx.x;
	const int tid = blockIdx.x;
	char *flag_head = NULL;
	if(bid == 0 && tid == 0){
		get_flag_head(table_type, &flag_head);
		mark_slot_free(flag_head, record_id);	
	}
}

__device__
void insert_rec(int table_type, void *record){
	int record_size = 0;
	int slot_id = -1;
	const int bid = blockIdx.x;
	const int tid = threadIdx.x;
	void *table_head = NULL;
	char *flag_head = NULL;
	if(bid == 0 && tid ==0){
		get_table_head(table_type, &table_head);
		get_record_size(table_type, &record_size);
		get_flag_head(table_type, &flag_head);
		slot_id = get_free_slot(flag_head, table_type);
		d_memcpy( table_head+slot_id*record_size, record, record_size);
		mark_slot_used(flag_head, slot_id);
	}
}

__device__
void update(int table_type, int record_id, void *record){
	void *table_head = NULL;
	int record_size;
	const int bid = blockIdx.x;
	const int tid = threadIdx.x;
	if( bid==0 && tid ==0){
		get_table_head(table_type, &table_head);
		get_record_size(table_type, &record_size);
		d_memcpy( table_head+record_id*record_size, record, record_size);
	}
}

__device__
void *get(int table_type, int record_id){
	void *table_head = NULL;
	int record_size;
	const int bid = blockIdx.x;
	const int tid = threadIdx.x;
	if(bid==0 && tid ==0){
		get_table_head(table_type, &table_head);
		get_record_size(table_type, &record_size);
		return (void *)(table_head+record_id*record_size);
	}
	return NULL;
}

// desc: scan table by compare some attribute in the table from the r_id.
// output: the record_id or -1 if there is no record match condition.
__device__
int table_scan(int table_type, int attr_type, int attr_size, int attr_offset, int op, void *value, int r_id){
	const int bid = blockIdx.x;
	const int tid = threadIdx.x;
	void *table_head = NULL;
	char *flag_head = NULL;
	int table_size;
	int record_size;
	if(bid == 0 && tid == 0){
		get_table_head(table_type, &table_head);
		get_table_size(table_type, &table_size);
		get_record_size(table_type, &record_size);
		get_flag_head(table_type, &flag_head);
		int i;
		for(i = r_id; i<table_size; i++){
			if(!(int)flag_head[i]){
				continue;
			}
			if(op == NO)
				return i;
			void *record_addr = (void *)(table_head + record_size*i);
			void *attr_addr = (void *)(record_addr + attr_offset);
			switch(attr_type){
				case(INT):
					{
					int des = *((int *)attr_addr);
					int src = *((int *)value);
					switch(op){
						case(EQ):
							if(des == src)
								return i;
							break;
						case(LT):
							if(des < src)
								return i;
							break;
						case(GT):
							if(des > src)
								return i;
							break;
						case(LE):
							if(des <= src)
								return i;
							break;
						case(GE):
							if(des >= src)
								return i;
							break;
						case(NE):
							if(des != src)
								return i;
							break;
					}
					}
					break;
				case(LONG):
					{
					long des = *((long *)attr_addr);
					long src = *((long *)value);
					switch(op){
						case(EQ):
							if(des == src)
								return i;
							break;
						case(LT):
							if(des < src)
								return i;
							break;
						case(GT):
							if(des > src)
								return i;
							break;
						case(LE):
							if(des <= src)
								return i;
							break;
						case(GE):
							if(des >= src)
								return i;
							break;
						case(NE):
							if(des != src)
								return i;
							break;
					}
					}
					break;
				case(DOUBLE):
					{
					double des = *((double *)attr_addr);
					double src = *((double *)value);
					switch(op){
						case(EQ):
							if(des == src)
								return i;
							break;
						case(LT):

							if(des < src)
								return i;
							break;
						case(GT):

							if(des > src)
								return i;
							break;
						case(LE):
							if(des <= src)
								return i;

							break;
						case(GE):
							if(des >= src)
								return i;
							break;
						case(NE):
							if(des != src)
								return i;
							break;
					}
					}
					break;
				case(STR):
					{
					char des[200];
					d_memcpy(des, (char *)attr_addr, attr_size);
					des[attr_size+1] = '\0';
					char src[200];
					d_memcpy(src, (char *)value, attr_size);
					src[attr_size+1] = '\0';
					switch(op){
						case(EQ):
							if( !d_strcmp(des, src) )
								return i;
							break;
						case(NE):
							if(d_strcmp(des, src))
								return i;
							break;
					}
					}
					break;
			}	
		}
	}	
	return -1;
}

/*
__device__
void get_next_record(){

}

__device__
void clsoe_scan(){

}
*/
__device__
void get_table_head(int table_type, void **table_head){

	switch(table_type){
		case(WAREHOUSE):
			*table_head = (void *)d_warehouses;
			break;
		case(STOCK):
			*table_head = (void *)d_stocks;
			break;
		case(DISTRICT):
			*table_head = (void *)d_districts;
			break;
		case(ITEM):
			*table_head = (void *)d_items;
			break;
		case(NEW_ORDER):
			*table_head = (void *)d_new_orders;
			break;
		case(ORDER):
			*table_head = (void *)d_orders;
			break;
		case(ORDER_LINE):
			*table_head = (void *)d_orderlines;
			break;
		case(CUSTOMER):
			*table_head = (void *)d_customers;
			break;
	}

}

__device__
void get_flag_head(int table_type, char **flag_head){

	switch(table_type){
		case(WAREHOUSE):
			*flag_head = d_warehouses_flag;
			break;
		case(STOCK):
			*flag_head = d_stocks_flag;
			break;
		case(DISTRICT):
			*flag_head = d_districts_flag;
			break;
		case(ITEM):
			*flag_head = d_items_flag;
			break;
		case(NEW_ORDER):
			*flag_head = d_new_orders_flag;
			break;
		case(ORDER):
			*flag_head = d_orders_flag;
			break;
		case(ORDER_LINE):
			*flag_head = d_orderlines_flag;
			break;
		case(CUSTOMER):
			*flag_head = d_customers_flag;
			break;
	}
}

__device__
void get_record_size(int table_type, int *record_size){
	
	switch(table_type){
		case(WAREHOUSE):
			*record_size = sizeof(struct warehouse);
			break;
		case(STOCK):
			*record_size = sizeof(struct stock);
			break;
		case(DISTRICT):
			*record_size = sizeof(struct district);
			break;
		case(ITEM):
			*record_size = sizeof(struct customer);
			break;
		case(NEW_ORDER):
			*record_size = sizeof(struct new_order);
			break;
		case(ORDER):
			*record_size = sizeof(struct order);
			break;
		case(ORDER_LINE):
			*record_size = sizeof(struct order_line);
			break;
		case(CUSTOMER):
			*record_size = sizeof(struct customer);
			break;
	}
}

__device__
void get_table_size(int table_type, int *table_size){

	switch(table_type){
		case(WAREHOUSE):
			*table_size = MAX_WAREHOUSE_NUM;
			break;
		case(STOCK):
			*table_size = MAX_STOCK_NUM;
			break;
		case(DISTRICT):
			*table_size = MAX_DISTRICT_NUM;
			break;
		case(ITEM):
			*table_size = MAX_ITEM_NUM;
			break;
		case(NEW_ORDER):
			*table_size = MAX_NEWORDER_NUM;
			break;
		case(ORDER):
			*table_size = MAX_ORDER_NUM;
			break;
		case(ORDER_LINE):
			*table_size = MAX_ORDER_LINE_NUM;
			break;
		case(CUSTOMER):
			*table_size = MAX_CUSTOMER_NUM;
			break;
	}
}
__device__
void  d_memcpy(void *des, void *src, int size){
	int i ;
	for(i = 0; i< size; i++){
		((char *)des)[i] = ((char *)src)[i];
	} 
}

__device__
int d_strcmp(char *des, char *src){
	int i=0;
	while(des[i] == src[i]){
		if(des[i] == '\0')	
			return 0;
		else{
			i++;
		}
	}
	return i;
}
