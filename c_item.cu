#include <stdio.h>  
#include <string.h>  
#include <stdlib.h>

#define IT_MAX_SIZE 100010 
#define IT_MAX_LENGTH 1000
#include <stdlib.h>
#include "tpcc_table.h"
int get_item(struct item *p_item_arr){ //?????????????????????????????????
	char file_name[] = "./csv/c_item.csv";  //?????????????????????????????????
    FILE *fp;  
    fp = fopen(file_name, "r");  
      
    if (!fp) {  
        fprintf(stderr, "failed to open file for reading\n");  
        return -1;  
    }
    
    char c;
    int i=0;
    int index=0;
    
    char **record_arr;
    record_arr=(char **)malloc(IT_MAX_SIZE*sizeof(char *));
    record_arr[index]=(char *)malloc(IT_MAX_LENGTH*sizeof(char));
    
    while(1){
      c=fgetc(fp);
      if(c==EOF){
        break;
      }
      if(c=='\n'){
        record_arr[index][i++]='\0';
        index++;
        record_arr[index]=(char *)malloc(IT_MAX_LENGTH*sizeof(char));
        i=0;
      }else{
        if(c!='"'){
          record_arr[index][i++]=c;
        }
      }
    }

//    *p_item_arr=(struct item *)malloc((index-1)*sizeof(struct item));//?????????????????????????????????
    
    for(i=1;i<index;i++){
      //printf("%s\n",record_arr[i]);
      char *token;
      
      token=strtok(record_arr[i],",");
      (p_item_arr)[i-1].I_ID=atol(token);//?????????????????????????????????
      //if(i<20) printf("I_ID: %ld, ", p_item_arr[i-1].I_ID);
      
      token=strtok(NULL, ",");
      (p_item_arr)[i-1].I_IM_ID=atol(token);//?????????????????????????????????
      //if(i<20) printf("I_IM_ID: %ld, ", p_item_arr[i-1].I_IM_ID);
      token=strtok(NULL, ",");
      strcpy((p_item_arr)[i-1].I_NAME,token);
      //if(i<20) printf("I_NAME: %s, ", p_item_arr[i-1].I_NAME);

      
	    token=strtok(NULL, ",");
      (p_item_arr)[i-1].I_PRICE=atof(token);//
      //if(i<20) printf("I_PRICE: %lf, ", p_item_arr[i-1].I_PRICE);
	  
      token=strtok(NULL, ",");
      strcpy((p_item_arr)[i-1].I_DATA,token);
      //if(i<20) printf("I_DATA: %s\n", p_item_arr[i-1].I_DATA);
      
      token=strtok(NULL, ",");
     	free(record_arr[i-1]); 
      //printf("!!!!!!!!!!!!!!!!\n%ld\n%s\n%s\n%s\n%s\n%s\n%s\n%lf\n%lf\n",item_arr[i].W_ID,item_arr[i].W_NAME,
	  //		item_arr[i].W_STREET_1,item_arr[i].W_STREET_2,item_arr[i].W_CITY,item_arr[i].W_STATE,
	  //		item_arr[i].W_ZIP,item_arr[i].W_TAX,item_arr[i].W_YTD);
    }
	free(record_arr);
     
    fclose (fp);  
    
    return index-1;
}
/*
int main(int argc, const char * argv[]) {  
    
    struct item *item_arr;//?????????????????????????????????
    int index=get_item(&item_arr);//?????????????????????????????????
    int i;
    for(i=1;i<index;i++){ //?????????????????????????????????
		printf("!!!!!!!!!!!!!!!!\n%ld\n %ld\n %s\n %lf\n %s\n",
        item_arr[i].I_ID,
        item_arr[i].I_IM_ID,
        item_arr[i].I_NAME,
        item_arr[i].I_PRICE,
        item_arr[i].I_DATA);
    }
    return 0;  
} */ 
