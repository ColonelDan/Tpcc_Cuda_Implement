#include <stdio.h>

#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>

#include "tpcc_table.h"
#include "tx.h"
#include "utility.h"
#include "table_operator.h"

//test for new-order transaction!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!

__device__
void tx_new_order(){
	//输入的数据
	long no_W_ID=1;	//warehouse ID
	long no_D_W_ID=1;	long no_D_ID=3;	//district ID
	long no_C_W_ID=1;	long no_C_D_ID=3;	long no_C_ID=3;	//customer ID
	int no_OL_CNT=2;	//order_line count
	long no_OL_I_ID_arr[] = {1,2};	//
	long no_OL_SUPPLY_W_ID_arr[] = {1,1};	//
	double no_OL_QUANTITY_arr[] = {1.0,2.0};

	long no_O_ID=1;//必须要用到的新订单的ID
	double no_OL_AMOUNT_arr[2];//必须要用到的各分录的价格
	long no_OL_NUMBER_arr[]={1,2};//必须要用到的各分录的分录号

	//事务启动
	printf("New-Order Transaction is start!\n");
	
	//在仓库表（warehouses）中查询仓库代码（W_ID）。取税率（W_TAX）。
	struct warehouse warehouse_tmp;	//为了求attr_offset 
	int rid_in_warehouses=0;
	//int table_scan(int table_type, int attr_type, int attr_size, int attr_offset, int op, void *value, int r_id);
	rid_in_warehouses= table_scan(WAREHOUSE, LONG, sizeof(long int), ((long)&(warehouse_tmp.W_ID)-(long)&(warehouse_tmp)), EQ, &no_W_ID, rid_in_warehouses);	//扫描整张warehouses表找出W_ID为no_W_ID的记录号 
	if(rid_in_warehouses==-1){	//warehouses中没有no_W_ID对应的记录 
		printf("Transaction Exception:\tW_ID is not in warehouse table!");
		return;
	}
	struct warehouse *p_warehouse_tmp =(struct warehouse *)get(WAREHOUSE, rid_in_warehouses);	//获取no_W_ID对应的记录
	printf("rid_in_warehouses:%d\tW_ID:%ld\tW_TAX:%lf\n",
		rid_in_warehouses,
		p_warehouse_tmp->W_ID,
		p_warehouse_tmp->W_TAX);//取税率(W_TAX)
	
	
	//在地区表（districts）中查询地区代码（D_W_ID,D_ID）。取税率（D_TAX）;取下一订单号（D_NEXT_O_ID）并且对其加一
	struct district district_tmp;
	struct district *p_district_tmp;
	int rid_in_districts=0;
	while(1){
		rid_in_districts = table_scan(DISTRICT, LONG, sizeof(long int), ((long)&(district_tmp.D_W_ID)-(long)&(district_tmp.D_ID)), EQ, &no_D_W_ID, rid_in_districts);	//扫描整张districts表找出D_W_ID为no_D_W_ID的记录号
		if(rid_in_districts == -1){	//districts中没有no_D_W_ID对应的记录 
			printf("Transaction Exception:\tD_W_ID is not in district table!");
			return;
		}
		p_district_tmp=(struct district *)get(DISTRICT, rid_in_districts);	//获取no_D_W_ID对应的记录
		if(p_district_tmp->D_ID == no_D_ID){	//district中有no_D_W_ID,no_D_ID对应的记录
			break;
		}else{
			rid_in_districts++;
		}
	}
	printf("rid_in_districts:%d\tD_W_ID:%ld\tD_ID:%ld\tD_TAX:%lf\tD_NEXT_O_ID:%ld\n",
		rid_in_districts,
		p_district_tmp->D_W_ID,
		p_district_tmp->D_ID,
		p_district_tmp->D_TAX,	//取税率（D_TAX）
		p_district_tmp->D_NEXT_O_ID);	//取下一订单号（D_NEXT_O_ID）
	//void  d_memcpy(void *des, void *src, int size)
	d_memcpy(&district_tmp,p_district_tmp,sizeof(struct district));
	district_tmp.D_NEXT_O_ID++;
	//void update(int table_type, int record_id, void *record);
	update(DISTRICT,rid_in_districts,&district_tmp);	//对下一订单号（D_NEXT_O_ID）加一
	// //验证D_NEXT_O_ID是否已经加一 
	// printf("rid_in_districts:%d\tD_W_ID:%ld\tD_ID:%ld\tD_NEXT_O_ID:%ld\n",
	// 	rid_in_districts,
	// 	p_district_tmp->D_W_ID,
	// 	p_district_tmp->D_ID,
	// 	p_district_tmp->D_NEXT_O_ID);	//取下一订单号（D_NEXT_O_ID）
	

	//在客户表（customers）中查询地区代码（C_W_ID,C_D_ID,C_ID）。取客户折扣率（C_DISCOUNT），姓（C_LAST），信用（C_CREDIT）
	struct customer customer_tmp;
	struct customer *p_customer_tmp;
	int rid_in_customers=0;
	while(1){
		rid_in_customers = table_scan(CUSTOMER, LONG, sizeof(long int), ((long)&(customer_tmp.C_W_ID)-(long)&(customer_tmp.C_ID)), EQ, &no_C_W_ID, rid_in_customers);	//扫描整张customer表找出C_W_ID为no_C_W_ID的记录号
		if(rid_in_customers == -1){	//customers中没有no_C_W_ID对应的记录 
			printf("Transaction Exception:\tC_W_ID is not in customer table!\n");
			return;
		}
		p_customer_tmp=(struct customer *)get(CUSTOMER, rid_in_customers);	//获取no_C_W_ID对应的记录
		if(p_customer_tmp->C_D_ID == no_C_D_ID && p_customer_tmp->C_ID == no_C_ID){	//customers中有no_C_W_ID,no_C_D_ID,no_C_ID对应的记录
			break;
		}else{
			rid_in_customers++;
		}
	}
	printf("rid_in_customers:%d\tC_W_ID:%ld\tC_D_ID:%ld\tC_ID:%ld\tC_DISCOUNT:%lf\tC_LAST:%s\tC_CREDIT:%s\n",
		rid_in_customers,
		p_customer_tmp->C_W_ID,
		p_customer_tmp->C_D_ID,
		p_customer_tmp->C_ID,
		p_customer_tmp->C_DISCOUNT,	//取客户折扣率（C_DISCOUNT）
		p_customer_tmp->C_LAST,	//姓（C_LAST）
		p_customer_tmp->C_CREDIT);	//信用（C_CREDIT）


	//向新订单表（new_orders）和订单表（orders）分别插入一条新记录
	int i;
	int no_O_ALL_LOCAL=0;
	for(i=0;i<no_OL_CNT;i++){
		if(no_W_ID!=no_OL_SUPPLY_W_ID_arr[i]){
			break;
		}
	}
	if(i==no_OL_CNT){
		no_O_ALL_LOCAL=1;
	}
	struct new_order new_order_tmp={no_O_ID, no_D_ID, no_W_ID};
	struct order order_tmp={no_O_ID, no_D_ID, no_W_ID, no_C_ID, 1189, 0, no_OL_CNT, no_O_ALL_LOCAL};
	insert_rec(NEW_ORDER,&new_order_tmp);
	int rid_in_orders=insert_rec(ORDER,&order_tmp);
	struct order *p_order_tmp=(struct order *)get(ORDER, rid_in_orders);
	printf("rid_in_orders:%d\tO_ID:%ld\tO_D_ID:%ld\tO_W_ID:%ld\tO_C_ID:%ld\tO_ENTRY_DATE:%ld\tO_CARRIER_ID:%ld\tO_OL_CNT:%d\tO_ALL_LOCAL:%d\n",
		rid_in_orders,
		p_order_tmp->O_ID,
		p_order_tmp->O_D_ID,
		p_order_tmp->O_W_ID,
		p_order_tmp->O_C_ID,
		p_order_tmp->O_ENTRY_DATE,
		p_order_tmp->O_CARRIER_ID,
		p_order_tmp->O_OL_CNT,
		p_order_tmp->O_ALL_LOCAL);

	//处理每笔分录（order_line）
	for(i=0;i<no_OL_CNT;i++){
		//在商品表（items）中查询商品代码（I_ID）。取价格（I_PRICE）、名称（I_NAME）、备注（I_DATA）
		struct item item_tmp;
		struct item *p_item_tmp;
		int rid_in_items=0;
		rid_in_items = table_scan(ITEM, LONG, sizeof(long int), ((long)&(item_tmp.I_ID)-(long)&(item_tmp.I_ID)), EQ, &no_OL_I_ID_arr[i], 0);	//扫描整张item表找出I_ID为no_OL_I_ID_arr[i]的记录号
		if(rid_in_items == -1){	//items中没有no_OL_I_ID_arr[i]对应的记录 
			printf("Transaction Exception:\tI_ID is not in item table!\n");
			return;
		}
		p_item_tmp=(struct item *)get(ITEM, rid_in_items);	//获取no_OL_I_ID_arr[i]对应的记录
		printf("rid_in_items:%d\tI_ID:%ld\tI_PRICE:%lf\tI_NAME:%s\tI_DATA:%s\n",
			rid_in_items,
			p_item_tmp->I_ID,
			p_item_tmp->I_PRICE,
			p_item_tmp->I_NAME,
			p_item_tmp->I_DATA);


		//在库存表（stocks）中查询商品代码（S_I_ID）与仓库代码（S_W_ID）。取库存（S_QUANTITY）、地区说明（S_DIST_XX）、备注（S_DATA）；
		struct stock stock_tmp;
		struct stock *p_stock_tmp;
		int rid_in_stocks=0;
		while(1){
			rid_in_stocks = table_scan(STOCK, LONG, sizeof(long int), ((long)&(stock_tmp.S_I_ID)-(long)&(stock_tmp.S_I_ID)), EQ, &no_OL_I_ID_arr[i], rid_in_stocks);	//扫描整张stock表找出S_I_ID为no_OL_I_ID_arr[i]的记录号
			if(rid_in_stocks == -1){	//stocks中没有no_OL_I_ID_arr[i]对应的记录 
				printf("Transaction Exception:\tS_I_ID is not in stock table!\n");
				return;
			}
			p_stock_tmp=(struct stock *)get(STOCK, rid_in_stocks);	//获取对应no_OL_I_ID_arr[i]的记录
			if(p_stock_tmp->S_W_ID == no_OL_SUPPLY_W_ID_arr[i]){	//stocks中有no_OL_I_ID_arr[i],no_OL_SUPPLY_W_ID_arr[i]对应的记录
				break;
			}else{
				rid_in_stocks++;
			}
		}
		char *no_S_DIST_tmp;
		switch(no_OL_SUPPLY_W_ID_arr[i]){
			case 1:no_S_DIST_tmp=p_stock_tmp->S_DIST_01;break;
			case 2:no_S_DIST_tmp=p_stock_tmp->S_DIST_02;break;
			case 3:no_S_DIST_tmp=p_stock_tmp->S_DIST_03;break;
			case 4:no_S_DIST_tmp=p_stock_tmp->S_DIST_04;break;
			case 5:no_S_DIST_tmp=p_stock_tmp->S_DIST_05;break;
			case 6:no_S_DIST_tmp=p_stock_tmp->S_DIST_06;break;
			case 7:no_S_DIST_tmp=p_stock_tmp->S_DIST_07;break;
			case 8:no_S_DIST_tmp=p_stock_tmp->S_DIST_08;break;
			case 9:no_S_DIST_tmp=p_stock_tmp->S_DIST_09;break;
			case 10:no_S_DIST_tmp=p_stock_tmp->S_DIST_10;break;
		}
		printf("rid_in_stocks:%d\tS_I_ID:%ld\tS_W_ID:%ld\tS_QUANTITY:%lf\tS_DIST_INFO:%s\tS_DATA:%s\n",
			rid_in_stocks,
			p_stock_tmp->S_I_ID,
			p_stock_tmp->S_W_ID,
			p_stock_tmp->S_QUANTITY,	//取库存（S_QUANTITY）
			no_S_DIST_tmp,	//地区说明（S_DIST_XX）
			p_stock_tmp->S_DATA);//	备注（S_DATA）
		d_memcpy(&stock_tmp,p_stock_tmp,sizeof(struct stock));
		stock_tmp.S_QUANTITY-=no_OL_QUANTITY_arr[i];	//库存量减去商品数量
		stock_tmp.S_YTD+=no_OL_QUANTITY_arr[i];	//累计供货数量加上商品数量
		stock_tmp.S_ORDER_CNT++;	//累计定单数量加一
		if(stock_tmp.S_W_ID!=no_W_ID){	//如果分录中的商品在其他仓库，则累计其他仓库供货数量加一
			stock_tmp.S_REMOTE_CNT++;
		}
		update(STOCK,rid_in_stocks,&stock_tmp);
		// //验证S_QUANTITY是否已经变化
		// printf("rid_in_stocks:%d\tS_I_ID:%ld\tS_W_ID:%ld\tS_QUANTITY:%lf\n",
		// 	rid_in_stocks,
		// 	p_stock_tmp->S_I_ID,
		// 	p_stock_tmp->S_W_ID,
		// 	p_stock_tmp->S_QUANTITY);//	取库存（S_QUANTITY）


		no_OL_AMOUNT_arr[i]=no_OL_QUANTITY_arr[i]*p_item_tmp->I_PRICE;//计算价格


		struct order_line new_orderline;
		new_orderline.OL_O_ID=no_O_ID,
		new_orderline.OL_D_ID=no_D_ID,
		new_orderline.OL_W_ID=no_W_ID,
		new_orderline.OL_NUMBER=no_OL_NUMBER_arr[i],
		new_orderline.OL_I_ID=no_OL_I_ID_arr[i],
		new_orderline.OL_SUPPLY_W_ID=no_OL_SUPPLY_W_ID_arr[i],
		new_orderline.OL_DELIVERY_D=0,
		new_orderline.OL_QUANTITY=no_OL_QUANTITY_arr[i],
		new_orderline.OL_AMOUNT=no_OL_AMOUNT_arr[i],
		d_memcpy(&new_orderline,no_S_DIST_tmp,24+1);
		insert_rec(ORDER_LINE,&new_orderline);	//分录表中插入新分录
	}

	double sum_OL_AMOUNT=0;
	for(i=0;i<no_OL_CNT;i++){
		sum_OL_AMOUNT+=no_OL_AMOUNT_arr[i];
	}
	double final_price=sum_OL_AMOUNT*(1-p_customer_tmp->C_DISCOUNT)*(1+p_warehouse_tmp->W_TAX+p_district_tmp->D_TAX);//计算定单总价格
	printf("final price is %lf\n",final_price);

	//事务提交 
	printf("New-Order Transaction is finished!\n");
}