#include "hip/hip_runtime.h"
//
//  desc: tpcc benchmark implementation in GPU
//	date: 2018-3-27
//	author: Xie Shangwei
//
#include <time.h>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include "utility.h"

// is same to memcpy.
__device__
void  d_memcpy(void *des, void *src, int size){
	int i ;
	for(i = 0; i< size; i++){
		((char *)des)[i] = ((char *)src)[i];
	} 
}


// is same to strcmp.
__device__
int d_strcmp(char *des, char *src){
	int i=1;
	while(des[i-1] == src[i-1]){
		if(des[i-1] == '\0')	
			return 0;
		else{
			i++;
		}
	}
	return i;
}


//get the des string length.
__device__
int d_strlen(char *des){
	int i = 0;
	while(des[i] != 0)
		i++;
	return i;
}

//generate randon number in [min, max)
__device__
unsigned int d_random(int min, int max){
	hiprandState state;
	int id = threadIdx.x;
	time_t t = clock();
	hiprand_init( (unsigned int)t, id, 0, &state);
	return hiprand(&state)%(max-min)+min;
}
