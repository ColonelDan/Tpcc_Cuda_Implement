#include "hip/hip_runtime.h"
//
//  desc: tpcc benchmark implementation in GPU
//	date: 2018-3-27
//	author: Xie Shangwei
//
#include <time.h>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include "utility.h"

// is same to memcpy.
__device__
void  d_memcpy(void *des, void *src, int size){
	int i ;
	for(i = 0; i< size; i++){
		((char *)des)[i] = ((char *)src)[i];
	} 
}


// is same to strcmp.
__device__
int d_strcmp(char *des, char *src){
	int i=1;
	while(des[i-1] == src[i-1]){
		if(des[i-1] == '\0')	
			return 0;
		else{
			i++;
		}
	}
	return i;
}

//get the des string length.
__device__
int d_strlen(char *des){
	int i = 0;
	while(des[i] != 0)
		i++;
	return i;
}

//str1=str1+str2, the limit is the size of str1(not length)
__device__
void d_str_append(char *str1,char *str2,int limit){
	int str1_length=d_strlen(str1);
	int str1_index=str1_length;
	int str2_index=0;
	while(str1_index!=limit-1 && str2[str2_index]!='\0'){
		str1[str1_index]=str2[str2_index];
		str1_index++;
		str2_index++;
	}
	str1[str1_index]='\0';
}

//is same to ltoa but not return result, as an alternative, the result is to be a parameter of the function
__device__
void d_ltoa(long n,char *s){
	int i,count,sign;
	if((sign=n)<0){//记录符号
		n=-n;//使n成为正数
	}
	count=0;
	do{
		s[count++]=n%10+'0';//取下一个数字
	}while ((n/=10)>0);//删除该数字
	if(sign<0){
		s[count++]='-';
	}
	char c;
	for(i=0;i<count/2;i++){//生成数字是逆序的，所以要头尾置换
		c=s[i];
		s[i]=s[count-1-i];
		s[count-1-i]=c;
	}
	s[count]='\0';
}

//generate randon number in [min, max)
__device__
unsigned int d_random(int min, int max){
	hiprandState state;
	int id = threadIdx.x;
	time_t t = clock();
	hiprand_init( (unsigned int)t, id, 0, &state);
	return hiprand(&state)%(max-min)+min;
}
