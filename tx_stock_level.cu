

#include  <stdio.h>

#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>

#include "tpcc_table.h"
#include "tx.h"
#include "utility.h"
#include "table_operator.h"

__device__
int d_max(int a, int b){
		return a>b?a:b;
}

__device__
void tx_stock_level(){
	printf("into stock_level\n");
	
	//generate parameters.
	long D_W_ID;
	long D_ID;
	long W_ID;
	unsigned int limit;

	unsigned int rid = d_random(0, 20);
	struct district tmp_district;
	rid = table_scan(DISTRICT, LONG, 0, 0, NO, NULL, rid);
	//printf("rid : %d\n", rid);
	void *content = get(DISTRICT, rid);
	d_memcpy(&tmp_district, content, sizeof(struct district));
	D_W_ID = tmp_district.D_W_ID;
	D_ID = tmp_district.D_ID;
	W_ID = tmp_district.D_W_ID;
	limit = d_random(30, 60);

	printf("****** stock_level ******\nparameters:\n W_ID : %ld\n D_W_ID : %ld\n D_ID : %ld\n limit : %u\n", D_W_ID, D_W_ID, D_ID, limit);

	int offset_D_W_ID = (unsigned int)&tmp_district.D_W_ID - (unsigned int)&tmp_district.D_ID;
	int offset_D_ID = 0;
	// int rid1 = table_scan(DISTRICT, LONG, 0, offset_D_W_ID, EQ, &D_W_ID, 0);
	// int rid2;
	// while( rid1 != -1){
	// 	rid2 = table_scan(DISTRICT, LONG, 0, offset_D_ID, EQ, &D_ID, rid);
	// 	if(rid1 == rid2)
	// 		break;
	// 	rid1 = table_scan(DISTRICT, LONG, 0, offset_D_W_ID, EQ, &D_W_ID, rid1+1);
	// }

	int start_id = 0;
	int id1 = table_scan(DISTRICT, LONG, 0, offset_D_W_ID, EQ, &D_W_ID, start_id);
	int id2 =  table_scan(DISTRICT, LONG, 0, offset_D_ID, EQ, &D_ID, start_id);
	while(id1 != id2){
		//printf("id1 : %d, id2:%d\n", id1, id2);
		if(id1 == -1 || id2 == -1){
			//printf("no record satisfy requirement\n");
			break;
		}
		start_id = d_max(id1, id2);
		id1 = table_scan(DISTRICT, LONG, 0, offset_D_W_ID, EQ, &D_W_ID, start_id);
		id2 = table_scan(DISTRICT, LONG, 0, offset_D_ID, EQ, &D_ID, start_id);
	}

	printf("*id1 : %d, id2:%d\n", id1, id2);
	if(id1 == id2&& id1 != -1)
			content = get(DISTRICT, id1);
	else
		printf("no record satisfy requirement\n");
	d_memcpy(&tmp_district, content, sizeof(struct district));

	long next_order_id = tmp_district.D_NEXT_O_ID;
	//printf("D_NEXT_ID : %ld\n", next_order_id);


	//next step
	// scan orderline where OL_W_ID = W_ID, OL_D_ID = D_ID, D_NEXT_O_ID-20 <= OL_O_ID < D_NEXT_O_ID.

	//struct order_line *orderline_result[20];
	struct order_line tmp_orderline;
	struct order_line *odl;
	long item_id[20];
	int offset_OL_W_ID = (int )&tmp_orderline.OL_W_ID - (int)&tmp_orderline.OL_O_ID;
	start_id = 0;
	//printf("offset_Ol_W_ID: %d\n", offset_OL_W_ID);
	int result_num = 0;
	int id = table_scan(ORDER_LINE, LONG, 0, offset_OL_W_ID, EQ, &W_ID, start_id);
	while(id != -1){
			odl = (struct order_line *)get(ORDER_LINE, id);
		 	if(odl->OL_D_ID == D_ID &&
		 		odl->OL_O_ID>=next_order_id-20 &&
		 		odl->OL_O_ID < next_order_id){
		 			item_id[result_num%20] = odl->OL_I_ID;
		 			result_num++;
					//printf("%ld 		%ld		%ld\n", odl->OL_O_ID, odl->OL_D_ID, odl->OL_W_ID);
		 	}
		 	id = table_scan(ORDER_LINE, LONG, 0, offset_OL_W_ID, EQ, &W_ID, id+1);
	}
	printf("OL_I_ID		\n");
	for(int i = 0; i<20; i++){
			printf("%ld\n", item_id[i]);
	}

	//next step.
	// scan stock where S_I_ID = OL_I_ID, S_W_ID=W_ID, S_QUANTITY < limit.
	start_id = 0;
	struct stock tmp_stock;
	int offset_S_I_ID = (unsigned int)&tmp_stock.S_I_ID - (unsigned int)&tmp_stock.
	id = table_scan(STOCK, LONG, )
}
