//
// desc: transaction delivery.
// date: 2018-4-8
// author: Xie Shangwei
//
//

#include <stdio.h>

#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>

#include "tpcc_table.h"
#include "tx.h"
#include "utility.h"
#include "table_operator.h"


__device__
void tx_delivery(){
	printf("transaction : delivery.\n");

	long D_ID;
	long W_ID;
	long D_W_ID;
	long O_CARRIER_ID;

	//generate parameters.
	int rid = d_random(0, 20);
	struct district *dis_p;
	rid = table_scan(DISTRICT, LONG, 0, 0, NO, NULL, rid);

	if(rid != -1)
		dis_p = (struct district *)get(DISTRICT, rid);
	else{
		printf("rid = -1, can not get the content.\n");
	}
	//dis_p = (struct district *)content;
	//printf("ok\n");
	//d_memcpy(&tmp_district, content, sizeof(struct district));
	D_W_ID = dis_p->D_W_ID;
	D_ID = dis_p->D_ID;
	W_ID = D_W_ID;
	O_CARRIER_ID = d_random(1, 11);

	printf("D_ID %ld\n", dis_p->D_ID);

	printf("parameters:\n");
	printf("D_W_ID : %ld\n", D_W_ID);
	printf("D_ID : %ld\n", D_ID);
	printf("W_ID : %ld\n", W_ID);
	printf("O_CARRIER_ID : %ld\n", O_CARRIER_ID);
	// transaction begin.
	// first step.
	struct new_order tmp_no;
	struct new_order *no_p;
	int offset_DID = (long)&tmp_no.NO_D_ID - (long)&tmp_no.NO_O_ID;
	int start_id = 0;
	rid = table_scan(NEW_ORDER, LONG , 0, offset_DID, EQ, &D_ID, start_id);
	while(rid != -1){
		no_p = (struct new_order *)get(NEW_ORDER, rid);
		if(no_p->NO_W_ID == W_ID){
			break;
		}
	}
	if(rid == -1)
		printf("there is no new_order satisfy condition\n");
	long NO_O_ID = no_p->NO_O_ID;
	printf("NO_O_ID : %ld\n", NO_O_ID);
	//return ;
	// delete record in the new_order table.
	struct order *or_p;
	start_id = 0;
	//int offset_O_D_ID = (long)&tmp_or.O_D_ID - (long)&tmp_or.O_ID;
	rid = table_scan(ORDER, LONG, 0, 0, EQ, &NO_O_ID, start_id);
	// printf("rid : %d\n", rid);
	// 	or_p = (struct order *)get(ORDER, rid);
	// 	printf(" wid : %ld , did : %ld, customer_id : %ld\n, ", or_p->O_W_ID, or_p->O_D_ID, or_p->O_C_ID);
	//return;
	while(rid != -1){
		or_p = (struct order *)get(ORDER, rid);
		if(or_p->O_W_ID == W_ID &&
			or_p->O_D_ID == D_ID ){
		//	printf("to be break\n");
			break;
		}
		//printf("wont break\n");
		rid = table_scan(ORDER, LONG, 0, 0, EQ, &NO_O_ID, rid+1);
		//printf("here rid = %ld\n", rid);
	}
	//printf("rid = %ld\n", rid);
	//return;
	if(rid == -1)
		printf("there is no record in table order satisfy condition.\n");
	long O_C_ID = or_p->O_C_ID;
	printf("O_C_ID : %ld\n", O_C_ID);
	//int offse_O_CARRIER_ID = (long)&tmp_or.O_CARRIER_ID - (long)&tmp_or.O_ID;
	//d_memcpy((void *)or_p+offse_O_CARRIER_ID, &O_CARRIER_ID, sizeof(long));
	or_p->O_CARRIER_ID = O_CARRIER_ID;
	printf("after set , carrier id is : %ld\n", (struct order *)or_p->O_CARRIER_ID);


	// next step operate in table order_line.
	start_id = 0;
	struct order_line *orl_p;
	//int offset_OL_W_ID = (long)&tmp_orl.OL_W_ID - (long)&tmp_orl.OL_O_ID;
	double sum_of_amout = 0;
	rid = table_scan(ORDER_LINE, LONG, 0, 0, EQ, &NO_O_ID, start_id);
	printf("order line rid = %ld\n", rid);
	//return;
	while(rid != -1){
		orl_p = (struct order_line *)get(ORDER_LINE, rid);
		if(orl_p->OL_D_ID == D_ID &&
			orl_p->OL_W_ID == W_ID ){
			orl_p -> OL_DELIVERY_D = 1234567;
			sum_of_amout += orl_p->OL_AMOUNT;
			//printf("here rid = %ld, amout : %lf\n", rid, orl_p->OL_AMOUNT);
		}
		rid = table_scan(ORDER_LINE, LONG, 0, 0, EQ, &NO_O_ID, rid+1);

	}
	printf("the OL_AMOUNT : %lf\n", sum_of_amout);
	//return;

	start_id = 0;
	struct customer *cus_p;
	//int offset_C_W_ID = (long)&cus.C_W_ID - (long)&cus.C_ID;
	rid = table_scan(CUSTOMER, LONG, 0, 0, EQ, &O_C_ID, start_id);
	//return;
	while(rid != -1){
		cus_p = (struct customer *)get(CUSTOMER, rid);
		if(cus_p->C_W_ID == W_ID &&
			cus_p->C_D_ID == D_ID){
			cus_p->C_BALANCE += sum_of_amout;
			cus_p->C_DELIVERY_CNT++;
			//printf("custmer rid : %d\n", rid);
			break;
		}
		rid = table_scan(CUSTOMER, LONG, 0, 0, EQ, &O_C_ID, rid+1);
	}
	if(rid == -1)
		printf("there is no record satisfy condition.\n");
	printf("customer id : %d\n", rid);
	printf("delivery process over!\n");
}